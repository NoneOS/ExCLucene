#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include </usr/include/sys/stat.h>
#include <errno.h>

#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <string>

#include <hip/hip_runtime.h>

#include "HS_SegLRC_Decompression_kernel.cu"

using namespace std;


string dataset_dir = "/media/indexDisk/naiyong/dataset/";
string data_dir = "/media/indexDisk/naiyong/data/HS_SegLRC/Compression/";
string regressioninfo_dir = "/media/indexDisk/naiyong/data/HS_SegLRC/Generator/";
string result_dir = "/media/indexDisk/naiyong/result/HS_SegLRC/Decompression/";

uint32_t kExceptedBucketSize = 256;


float FRAC = 0.0;
int step = 0;
ofstream ofsresult;


#define ITERATION 2
uint32_t batchID = 0;


unsigned char *ptr = NULL;       // pointer of queryset
at_search_ind_t *patind = NULL;  // pointer of struct for ind1 and ind2


#define BUFFER_SIZE 2048
unsigned char buffer[BUFFER_SIZE];
inline void readFile(unsigned char *ptr, FILE *fp) {
	uint64_t count = 0, bytes = 0;
	while ((bytes = fread(ptr+count, 1, BUFFER_SIZE, fp)) > 0) {
		count += bytes;
	}
}


void as_load_atind(const char *dbi_dir, const char *ind_name) {
	patind = (at_search_ind_t *)malloc(sizeof(*patind));
	struct stat buf;

	char file_name[MAX_PATH_LEN];
	patind->fd_ind1 = 0;
	patind->fd_ind2 = 0;
	patind->m_pind1 = 0;
	patind->m_pind2 = 0;

	// ind1
	sprintf (file_name, "%s%s.ind1", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind1 = buf.st_size;
	patind->m_tcount = buf.st_size / sizeof (at_term_ind1_t);
	patind->fd_ind1 = fopen(file_name, "rb");
	patind->m_pind1 = (at_term_ind1_t *)malloc(buf.st_size);
	readFile((unsigned char*)patind->m_pind1 , patind->fd_ind1);

	//ind2
	sprintf (file_name, "%s%s.ind2", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind2 = buf.st_size;
	patind->fd_ind2 = fopen(file_name,"rb");
	patind->m_pind2 = (unsigned char*) malloc(buf.st_size);
	readFile(patind->m_pind2 , patind->fd_ind2);
}


void allocateResource() {
	// host resources
	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_queryID_perBlock, MAX_BLOCK_NUM * sizeof(uint16_t)));

	// device resources
	CUDA_SAFE_CALL(hipMalloc((void**)&d_shortest_lists, baseSize));					
	CUDA_SAFE_CALL(hipMalloc((void**)&d_segoffset, segoffset_size));				
	CUDA_SAFE_CALL(hipMalloc((void**)&d_regression_info, regression_info_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_base, base_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_hash_info, hash_info_size));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_lists, patind->sz_ind2));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_queryID_perBlock, MAX_BLOCK_NUM * sizeof(uint16_t)));

	if (!(d_lists)) {
		cout << "allocation failed" << endl;
		exit(1);
	}


	// transfer lists
	h_lists = (uint32_t *)patind->m_pind2;
	CUDA_SAFE_CALL(hipMemcpy(d_lists, h_lists, patind->sz_ind2, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_segoffset, h_segoffset, segoffset_size, hipMemcpyHostToDevice));	

	CUDA_SAFE_CALL(hipMemcpy(d_regression_info, h_regression_info, regression_info_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_base, h_base, base_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_hash_info, h_hash_info, hash_info_size, hipMemcpyHostToDevice)); 
}



batchInfo CPUPreprocess(unsigned char *&ptr_in, const unsigned char *ptr_end) {
	batchInfo bi;			
	query_input_t input;
	testCell tc;	
	register uint32_t tcount = 0;	


	uint32_t current_blockNum;
	uint64_t ucntShortest_sum = 0; 
	uint32_t blockNum = 0;
	uint32_t nTotalQueryNum = 0;
	uint32_t queries_offset = 0;

	while (ucntShortest_sum <= Threshold && ptr_in < ptr_end) {
		input.tnum = *ptr_in;
		ptr_in += sizeof(uint32_t);
		memcpy(input.tno, ptr_in, sizeof(uint32_t) * input.tnum);
		ptr_in += sizeof(uint32_t) * input.tnum;

		tc.tcount = input.tnum;
		tcount = tc.tcount;
		for (uint32_t i = 0; i < tcount; ++i) {
			at_term_ind1_t *pind1 = patind->m_pind1 + input.tno[i];
			tc.ucnt[i] = pind1->m_urlcount;  
			tc.uwlist[i] = (pind1->m_off) / sizeof(uint32_t);
			tc.tno[i] = input.tno[i];
			tc.offset[i] = *(h_offset + input.tno[i]);
			tc.hashoffset[i] = *(h_hash_offset + input.tno[i]);
		}

		// insertion sort
		for (uint32_t i = 1; i < tcount; ++i) {
			uint32_t k = i;
			uint32_t uwlist_tmp = tc.uwlist[i];
			uint32_t ucnt_tmp = tc.ucnt[i];
			uint32_t tno_tmp = tc.tno[i];
			uint32_t offset_tmp = tc.offset[i];
			uint32_t hashoffset_tmp = tc.hashoffset[i];

			while (k && ucnt_tmp < tc.ucnt[k - 1]) {
				tc.uwlist[k] = tc.uwlist[k - 1];
				tc.ucnt[k] = tc.ucnt[k - 1];
				tc.tno[k] = tc.tno[k - 1];
				tc.offset[k] = tc.offset[k - 1];
				tc.hashoffset[k] = tc.hashoffset[k - 1];
				--k;
			};

			if (k != i)
			{
				tc.uwlist[k] = uwlist_tmp;
				tc.ucnt[k] = ucnt_tmp;
				tc.tno[k] = tno_tmp;
				tc.offset[k] = offset_tmp;
				tc.hashoffset[k] = hashoffset_tmp;
			}
		}

		// calculate block number needed by current query
		current_blockNum = tc.ucnt[0] % THREAD_NUM ? tc.ucnt[0] / THREAD_NUM + 1 : tc.ucnt[0] / THREAD_NUM;

		// set host memory for constant memory
		h_startBlockId[nTotalQueryNum] = blockNum;
		h_baseOffset[nTotalQueryNum] = ucntShortest_sum;
		h_queriesOffset[nTotalQueryNum] = queries_offset;
		

		// copy the query from tc to h_queries
		*(h_queries + queries_offset) = tc.tcount;
		*(h_queries + queries_offset + 1) = ucntShortest_sum;
		memcpy(h_queries + queries_offset + 2, tc.ucnt, sizeof(uint32_t) * tc.tcount);
		memcpy(h_queries + queries_offset + 2 + tc.tcount, tc.uwlist, sizeof(uint32_t) * tc.tcount);
		memcpy(h_queries + queries_offset + 2 + 2 * tc.tcount, tc.offset, sizeof(uint32_t) * tc.tcount);
		memcpy(h_queries + queries_offset + 2 + 3 * tc.tcount, tc.hashoffset, sizeof(uint32_t) * tc.tcount);
		// copy ends

		// set queryID for each block
		for (uint32_t k = 0; k < current_blockNum; ++k) {
			h_queryID_perBlock[blockNum + k] = nTotalQueryNum;	
		}

		//set several local variables for next loop
		blockNum += current_blockNum;
		ucntShortest_sum += tc.ucnt[0];
		queries_offset += tc.tcount * 4 + 2;
		++nTotalQueryNum;
	};

	
	
	if (baseSize / sizeof(uint32_t) < ucntShortest_sum) {
		cout << "ucntShortest_sum: " << ucntShortest_sum << "exceeds baseSize: " << baseSize << endl;
		exit(1);
	}

	if (MAX_BLOCK_NUM < blockNum) {
		cout << "blockNum is over " << MAX_BLOCK_NUM  << endl;;
		exit(1);
	}
	

	// prepare for return value
	bi.blockNum = blockNum;
	bi.constantUsedInByte = (nTotalQueryNum * 3 + queries_offset + batchInfoElementNum) * sizeof(uint32_t);  
	bi.ucntShortest_sum = ucntShortest_sum;
	bi.nTotalQueryNum = nTotalQueryNum;

	// integrate five arrays into h_constant
	memcpy(h_constant, &bi, sizeof(struct batchInfo));
	memcpy(h_constant + batchInfoElementNum, h_startBlockId, sizeof(uint32_t) * nTotalQueryNum);
	memcpy(h_constant + batchInfoElementNum + nTotalQueryNum, h_queriesOffset, sizeof(uint32_t) * nTotalQueryNum);
	memcpy(h_constant + batchInfoElementNum + nTotalQueryNum * 2, h_queries, queries_offset * sizeof(uint32_t));

	return bi;
}


void htodTransfer(const batchInfo &bi) {
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_constant), h_constant, bi.constantUsedInByte));
	CUDA_SAFE_CALL(hipMemcpy(d_queryID_perBlock, h_queryID_perBlock, bi.blockNum * sizeof(uint16_t), hipMemcpyHostToDevice));
}



void kernelInvoke(const batchInfo &bi) {
#ifdef debug	
	printf("blockNum:%u\n",bi.blockNum);
#endif

	HS_SegLRC_Decompression<<<bi.blockNum, THREAD_NUM>>>(d_lists, d_queryID_perBlock, d_shortest_lists, d_segoffset, d_regression_info, d_base, d_hash_info);
	CUDA_SAFE_CALL(hipDeviceSynchronize());


#ifdef debug
	if (batchID < 5) {
		// debug; transback h_isCommon and h_scan_odata
		FILE *fisCommon = fopen("slist", "a+");
		char isCommon[256];
		unsigned int *h_isCommon = (unsigned int*)malloc(baseSize);
		CUDA_SAFE_CALL(hipMemcpy(h_isCommon, d_shortest_lists, baseSize, hipMemcpyDeviceToHost));
		for (int i = 0; i < bi.ucntShortest_sum; ++i)
		{
			sprintf(isCommon, "batch:%u\ti:%u\t%u\n", batchID, i, h_isCommon[i]);
			fputs(isCommon, fisCommon);
			fflush(fisCommon);
		}
		free(h_isCommon);
		fclose(fisCommon);
		// debug ends
	}
#endif
}

void releaseResource() {
	free(h_offset);
	h_offset = NULL;

	free(h_segoffset);
	h_segoffset = NULL;

	free(h_regression_info);
	h_regression_info = NULL;

	free(h_base);
	h_base = NULL;

	free(h_hash_offset);
	h_hash_offset = NULL;

	free(h_hash_info);
	h_hash_info = NULL;

	CUDA_SAFE_CALL(hipHostFree(h_queryID_perBlock));
	h_queryID_perBlock = NULL;


	CUDA_SAFE_CALL(hipFree(d_shortest_lists));
	d_shortest_lists = NULL;

	CUDA_SAFE_CALL(hipFree(d_segoffset));
	d_segoffset = NULL;

	CUDA_SAFE_CALL(hipFree(d_regression_info));
	d_regression_info = NULL;

	CUDA_SAFE_CALL(hipFree(d_base));
	d_base = NULL;

	CUDA_SAFE_CALL(hipFree(d_hash_info));
	d_hash_info = NULL;

	CUDA_SAFE_CALL(hipFree(d_lists));
	d_lists = NULL;

	CUDA_SAFE_CALL(hipFree(d_queryID_perBlock));
	d_queryID_perBlock = NULL;
}


template <uint64_t beginThreshold>
void Run(unsigned char *ptr_in, unsigned char *ptr_end) {
	Threshold = beginThreshold;

	batchInfo bi;
	unsigned char *ptr_in_old = ptr_in, *ptr_end_old = ptr_end;

	hipEvent_t start, stop;
	float time_CPU = 0, time_htod = 0, time_kernel = 0;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	cout << "begin decompressing..." << endl;
	if (Threshold < 1024 * 1024) 
		cout << "Threshold: " << Threshold / 1024 << "K";
	else 
		cout << "Threshold: " << Threshold / (1024 * 1024) << "M";

	cout << ", THREAD_NUM: " << THREAD_NUM << endl;

	allocateResource();

	uint32_t nTotalQueryNum = 0;
	uint64_t ucntShortest_sum = 0;
	for (int iteration = 0; iteration < ITERATION; ++iteration) {
		ptr_in = ptr_in_old;
		ptr_end = ptr_end_old;

		batchID = 0;
		nTotalQueryNum = 0;
		ucntShortest_sum = 0;
		while (ptr_in < ptr_end) {
			hipEventRecord(start, 0);

			bi = CPUPreprocess(ptr_in, ptr_end);

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);
			time_CPU += elapsedTime;

			hipEventRecord(start, 0);

			htodTransfer(bi);

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);
			time_htod += elapsedTime;


			hipEventRecord(start, 0);

			kernelInvoke(bi);

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);
			time_kernel += elapsedTime;


			++batchID;
			nTotalQueryNum += bi.nTotalQueryNum;
			ucntShortest_sum += bi.ucntShortest_sum;
		}
	}

	time_CPU /= ITERATION;
	time_htod /= ITERATION;
	time_kernel /= ITERATION;

	cout << "number of queries: " << nTotalQueryNum << endl;
	cout << "number of batches: " << batchID << endl;
	cout << "ucntShortest_sum: " << ucntShortest_sum << endl;
	cout << "CPU time: " <<  time_CPU << "ms" << endl
		 << "htod time: " <<  time_htod << "ms" << endl
		 << "kernel time: " <<  time_kernel << "ms" << endl << endl;

	float fThroughput = ucntShortest_sum / time_kernel * 1000;
	fThroughput /= 1024 * 1024 * 1024;
	ofsresult << FRAC << "\t" << fThroughput << endl;


	releaseResource();


	hipEventDestroy(start);
	hipEventDestroy(stop);
}


void terminator(){
	if (ptr != NULL) 
		free(ptr);
	if (patind->m_pind1 != NULL) 
		free(patind->m_pind1);
	if (patind->m_pind2 != NULL) 
		free(patind->m_pind2);
	if (patind != NULL) 
		free(patind);

	ptr = NULL;
	patind->m_pind1 = NULL;
	patind->m_pind2 = NULL;
	patind = NULL;
}


void runTest(int argc, char **argv) {
	if (argc < 2) {
		cout << "wrong number of arguments" << endl;
		exit(1);
	}
	string dataset = argv[1];

	if (argc > 3) {
		CUDA_SAFE_CALL(hipSetDevice(strtol(argv[3], NULL, 10)));
	}
	else {
		CUDA_SAFE_CALL(hipSetDevice(0));
	}


	string queryset = dataset_dir + dataset + "/" + dataset + ".query";
	char index_dir[MAX_PATH_LEN];
	char offset_file[MAX_PATH_LEN];
	char segoffset_file[MAX_PATH_LEN];
	char regression_info_file[MAX_PATH_LEN];
	char base_file[MAX_PATH_LEN];
	char hash_offset_file[MAX_PATH_LEN];
	char hash_info_file[MAX_PATH_LEN];


	sprintf(index_dir, "%s%d/", data_dir.c_str(), step);
	sprintf(offset_file, "%s%d/%s.segNum", data_dir.c_str(), step, dataset.c_str());
	sprintf(segoffset_file, "%s%d/%s.segOffset", data_dir.c_str(), step, dataset.c_str());
	sprintf(regression_info_file, "%s%s%s", regressioninfo_dir.c_str(), dataset.c_str(), ".regression_info");
	sprintf(base_file, "%s%s%s", regressioninfo_dir.c_str(), dataset.c_str(), ".ind2_distance_base");
	sprintf(hash_offset_file, "%s%s%s", regressioninfo_dir.c_str(), dataset.c_str(), ".HashSeg_offset");
	sprintf(hash_info_file, "%s%s%s", regressioninfo_dir.c_str(), dataset.c_str(), ".HashSeg_info");


	struct stat buf;

	FILE *foffset = fopen(offset_file, "rb");
	if (!foffset) {
		cout << offset_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(offset_file, &buf);
	h_offset = (uint32_t *)malloc(buf.st_size);			
	readFile((unsigned char *)h_offset, foffset);
	fclose(foffset);

	FILE *fsegoffset = fopen(segoffset_file, "rb");
	if (!fsegoffset) {
		cout << segoffset_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(segoffset_file, &buf);
	segoffset_size = buf.st_size;							
	h_segoffset = (uint32_t *)malloc(segoffset_size);	
	readFile((unsigned char *)h_segoffset, fsegoffset);
	fclose(fsegoffset);

	FILE *fregression_info = fopen(regression_info_file, "rb");
	if (!fregression_info) {
		cout << regression_info_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(regression_info_file, &buf);
	regression_info_size = buf.st_size;
	h_regression_info = (regression_info_t*)malloc(regression_info_size);
	readFile((unsigned char *)h_regression_info, fregression_info);
	fclose(fregression_info);

	FILE *fbase = fopen(base_file, "rb");
	if (!fbase) {
		cout << base_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(base_file, &buf);
	base_size = buf.st_size;
	h_base = (int *)malloc(base_size);
	readFile((unsigned char *)h_base, fbase);
	fclose(fbase);

	
	FILE *fhash_offset = fopen(hash_offset_file, "rb");
	if (!fhash_offset) {
		cout << hash_offset_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(hash_offset_file, &buf);
	hash_offset_size = buf.st_size;
	h_hash_offset = (uint32_t*)malloc(hash_offset_size);
	readFile((unsigned char*)h_hash_offset, fhash_offset);
	fclose(fhash_offset);

	FILE *fhash_info = fopen(hash_info_file, "rb");
	if (!fhash_info) {
		cout << hash_info_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(hash_info_file, &buf);
	hash_info_size = buf.st_size;
	h_hash_info = (uint32_t*)malloc(hash_info_size);
	readFile((unsigned char*)h_hash_info, fhash_info);
	fclose(fhash_info);

	
	FILE *fquery = fopen(queryset.c_str(),"rb");
	if (!fquery) {
		cout << queryset << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(queryset.c_str(), &buf);  
	uint32_t querysize = buf.st_size;
	ptr = (unsigned char *)malloc(querysize);					
	readFile(ptr, fquery);
	fclose(fquery);


	as_load_atind(index_dir, dataset.c_str());
	
	Run<1024 * 1024>(ptr, ptr + querysize);


	terminator();


	CUDA_SAFE_CALL(hipDeviceReset());
}


int main(int argc, char** argv)  {
	if (argc < 2) {
		cout << "wrong number of arguments" << endl;
		exit(1);
	}
	string dataset = argv[1];

	if (argc > 2) {
		kExceptedBucketSize = atoi(argv[2]);
	}
	cout << "HS" << kExceptedBucketSize << "_SegLRC_Decompression" << endl;
	cout << "dataset = " << dataset << endl;


	ostringstream result;
	result << result_dir << dataset << "_HS" << kExceptedBucketSize << "_SegLRC_Decompression.txt";
	ofsresult.open((result.str()).c_str());
	ofsresult << "Threshold = 1M" << endl;
	ofsresult << "Throughput: G docIDs/s" << endl << endl;
	ofsresult << "FRAC\tThroughput" << endl;

	float FRAC_begin = 0.0;
	float FRAC_step = 0.04;
	float FRAC_end = 0.64;
	for (step = 0, FRAC = FRAC_begin; FRAC <= FRAC_end; FRAC += FRAC_step) {
		cout << "step = " << step << ", FRAC = " << FRAC << endl;
		runTest(argc, argv);
		++step;
	}

	ofsresult.close();

	return 0;
}
