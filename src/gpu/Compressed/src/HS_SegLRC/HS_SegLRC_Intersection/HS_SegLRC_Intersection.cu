#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include </usr/include/sys/stat.h>
#include <errno.h>

#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <string>

#include <cudpp.h>
#include <hip/hip_runtime.h>

#include "HS_SegLRC_Intersection_kernel.cu"

using namespace std;


string dataset_dir = "/media/indexDisk/naiyong/dataset/";
string data_dir = "/media/indexDisk/naiyong/data/HS_SegLRC/Compression/0/";
string regressioninfo_dir = "/media/indexDisk/naiyong/data/HS_SegLRC/Generator/";
string result_dir = "/media/indexDisk/naiyong/result/HS_SegLRC/Intersection/";

uint32_t kExceptedBucketSize = 256;

#define ITERATION 2
uint32_t batchID = 0;


unsigned char *ptr = NULL;       // pointer of queryset
at_search_ind_t *patind = NULL;  // pointer of struct for ind1 and ind2


#define BUFFER_SIZE 2048
unsigned char buffer[BUFFER_SIZE];
inline void readFile(unsigned char *ptr, FILE *fp) {
	uint64_t count = 0, bytes = 0;
	while ((bytes = fread(ptr+count, 1, BUFFER_SIZE, fp)) > 0) {
		count += bytes;
	}
}


void as_load_atind(const char *dbi_dir, const char *ind_name) {
	patind = (at_search_ind_t *)malloc(sizeof(*patind));
	struct stat buf;

	char file_name[MAX_PATH_LEN];
	patind->fd_ind1 = 0;
	patind->fd_ind2 = 0;
	patind->m_pind1 = 0;
	patind->m_pind2 = 0;

	// ind1
	sprintf (file_name, "%s%s.ind1", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind1 = buf.st_size;
	patind->m_tcount = buf.st_size / sizeof (at_term_ind1_t);
	patind->fd_ind1 = fopen(file_name, "rb");
	patind->m_pind1 = (at_term_ind1_t *)malloc(buf.st_size);
	readFile((unsigned char*)patind->m_pind1 , patind->fd_ind1);

	//ind2
	sprintf (file_name, "%s%s.ind2", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind2 = buf.st_size;
	patind->fd_ind2 = fopen(file_name,"rb");
	patind->m_pind2 = (unsigned char*) malloc(buf.st_size);
	readFile(patind->m_pind2 , patind->fd_ind2);
}


void allocateResource() {
	// host resources
	h_result = (uint32_t *)malloc(baseSize);
	h_ucntResult = (uint32_t *)malloc(5000 * sizeof(uint32_t));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_queryID_perBlock, MAX_BLOCK_NUM * sizeof(uint16_t)));

	// device resources
	CUDA_SAFE_CALL(hipMalloc((void**)&d_shortest_lists, baseSize));					
	CUDA_SAFE_CALL(hipMalloc((void**)&d_segoffset, segoffset_size));				
	CUDA_SAFE_CALL(hipMalloc((void**)&d_regression_info, regression_info_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_base, base_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_hash_info, hash_info_size));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_isCommon, baseSize));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_scan_odata, baseSize));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_result, baseSize));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_ucntResult, 5000 * sizeof(uint32_t)));	// no more than 5000 queries in a batch, otherwise the response time will be unacceptable
	CUDA_SAFE_CALL(hipMalloc((void**)&d_lists, patind->sz_ind2));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_queryID_perBlock, MAX_BLOCK_NUM * sizeof(uint16_t)));

	if (!(h_result && d_lists && d_isCommon && d_scan_odata && d_result)) {
		cout << "allocation failed" << endl;
		exit(1);
	}

	
	// transfer lists
	h_lists = (uint32_t *)patind->m_pind2;
	CUDA_SAFE_CALL(hipMemcpy(d_lists, h_lists, patind->sz_ind2, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_segoffset, h_segoffset, segoffset_size, hipMemcpyHostToDevice));	

	CUDA_SAFE_CALL(hipMemcpy(d_regression_info, h_regression_info, regression_info_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_base, h_base, base_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_hash_info, h_hash_info, hash_info_size, hipMemcpyHostToDevice)); 


	// cudpp alloc
	cudppCreate(&theCudpp);

	config.op = CUDPP_ADD;
	config.datatype = CUDPP_INT;
	config.algorithm = CUDPP_SCAN;
	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
	CUDPPResult result = cudppPlan(theCudpp, &scanplan, config, 40000000, 1, 0);

	if (CUDPP_SUCCESS != result) {
		cout << "cudpp allocationg falied" << endl;
		exit(1);
	}

}


batchInfo CPUPreprocess(unsigned char *&ptr_in, const unsigned char *ptr_end) {
	batchInfo bi;			
	query_input_t input;
	testCell tc;	
	register uint32_t tcount = 0;	


	uint32_t current_blockNum;
	uint64_t ucntShortest_sum = 0; 
	uint32_t blockNum = 0;
	uint32_t nTotalQueryNum = 0;
	uint32_t queries_offset = 0;

	while (ucntShortest_sum <= Threshold && ptr_in < ptr_end) {
		input.tnum = *ptr_in;
		ptr_in += sizeof(uint32_t);
		memcpy(input.tno, ptr_in, sizeof(uint32_t) * input.tnum);
		ptr_in += sizeof(uint32_t) * input.tnum;

		tc.tcount = input.tnum;
		tcount = tc.tcount;
		for (uint32_t i = 0; i < tcount; ++i) {
			at_term_ind1_t *pind1 = patind->m_pind1 + input.tno[i];
			tc.ucnt[i] = pind1->m_urlcount;  
			tc.uwlist[i] = (pind1->m_off) / sizeof(uint32_t);
			tc.tno[i] = input.tno[i];
			tc.offset[i] = *(h_offset + input.tno[i]);
			tc.hashoffset[i] = *(h_hash_offset + input.tno[i]);
		}

		// insertion sort
		for (uint32_t i = 1; i < tcount; ++i) {
			uint32_t k = i;
			uint32_t uwlist_tmp = tc.uwlist[i];
			uint32_t ucnt_tmp = tc.ucnt[i];
			uint32_t tno_tmp = tc.tno[i];
			uint32_t offset_tmp = tc.offset[i];
			uint32_t hashoffset_tmp = tc.hashoffset[i];

			while (k && ucnt_tmp < tc.ucnt[k - 1]) {
				tc.uwlist[k] = tc.uwlist[k - 1];
				tc.ucnt[k] = tc.ucnt[k - 1];
				tc.tno[k] = tc.tno[k - 1];
				tc.offset[k] = tc.offset[k - 1];
				tc.hashoffset[k] = tc.hashoffset[k - 1];
				--k;
			};

			if (k != i)
			{
				tc.uwlist[k] = uwlist_tmp;
				tc.ucnt[k] = ucnt_tmp;
				tc.tno[k] = tno_tmp;
				tc.offset[k] = offset_tmp;
				tc.hashoffset[k] = hashoffset_tmp;
			}
		}

		// calculate block number needed by current query
		current_blockNum = tc.ucnt[0] % THREAD_NUM ? tc.ucnt[0] / THREAD_NUM + 1 : tc.ucnt[0] / THREAD_NUM;

		// set host memory for constant memory
		h_startBlockId[nTotalQueryNum] = blockNum;
		h_baseOffset[nTotalQueryNum] = ucntShortest_sum;
		h_queriesOffset[nTotalQueryNum] = queries_offset;
		

		// copy the query from tc to h_queries
		*(h_queries + queries_offset) = tc.tcount;
		*(h_queries + queries_offset + 1) = ucntShortest_sum;
		memcpy(h_queries + queries_offset + 2, tc.ucnt, sizeof(uint32_t) * tc.tcount);
		memcpy(h_queries + queries_offset + 2 + tc.tcount, tc.uwlist, sizeof(uint32_t) * tc.tcount);
		memcpy(h_queries + queries_offset + 2 + 2 * tc.tcount, tc.offset, sizeof(uint32_t) * tc.tcount);
		memcpy(h_queries + queries_offset + 2 + 3 * tc.tcount, tc.hashoffset, sizeof(uint32_t) * tc.tcount);
		// copy ends

		// set queryID for each block
		for (uint32_t k = 0; k < current_blockNum; ++k) {
			h_queryID_perBlock[blockNum + k] = nTotalQueryNum;	
		}

		//set several local variables for next loop
		blockNum += current_blockNum;
		ucntShortest_sum += tc.ucnt[0];
		queries_offset += tc.tcount * 4 + 2;
		++nTotalQueryNum;
	};

	
	
	if (baseSize / sizeof(uint32_t) < ucntShortest_sum) {
		cout << "ucntShortest_sum: " << ucntShortest_sum << "exceeds baseSize: " << baseSize << endl;
		exit(1);
	}

	if (MAX_BLOCK_NUM < blockNum) {
		cout << "blockNum is over " << MAX_BLOCK_NUM  << endl;;
		exit(1);
	}
	

	// prepare for return value
	bi.blockNum = blockNum;
	bi.constantUsedInByte = (nTotalQueryNum * 3 + queries_offset + batchInfoElementNum) * sizeof(uint32_t);  
	bi.ucntShortest_sum = ucntShortest_sum;
	bi.nTotalQueryNum = nTotalQueryNum;

	// integrate five arrays into h_constant
	memcpy(h_constant, &bi, sizeof(struct batchInfo));
	memcpy(h_constant + batchInfoElementNum, h_startBlockId, sizeof(uint32_t) * nTotalQueryNum);
	memcpy(h_constant + batchInfoElementNum + nTotalQueryNum, h_queriesOffset, sizeof(uint32_t) * nTotalQueryNum);
	memcpy(h_constant + batchInfoElementNum + nTotalQueryNum * 2, h_queries, queries_offset * sizeof(uint32_t));

	return bi;
}


void htodTransfer(const batchInfo &bi) {
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_constant), h_constant, bi.constantUsedInByte));
	CUDA_SAFE_CALL(hipMemcpy(d_queryID_perBlock, h_queryID_perBlock, bi.blockNum * sizeof(uint16_t), hipMemcpyHostToDevice));
}


void kernelInvoke(const batchInfo &bi) {
#ifdef debug	
	printf("blockNum:%u\n",bi.blockNum);
#endif
	
	HS_SegLRC_Intersection<<<bi.blockNum, THREAD_NUM>>>(d_lists, d_isCommon, d_queryID_perBlock, d_shortest_lists, d_segoffset, d_regression_info, d_base, d_hash_info);
	CUDA_SAFE_CALL(hipDeviceSynchronize());


#ifdef debug
	if (batchID < 1)
	{
		//debug; transback h_isCommon and h_scan_odata
		FILE *fisCommon = fopen("isCommon", "a+");
		char isCommon[256];
		uint32_t *h_isCommon = (uint32_t*)malloc(baseSize);
		CUDA_SAFE_CALL(hipMemcpy(h_isCommon, d_isCommon, baseSize, hipMemcpyDeviceToHost));
		for (int i = 0; i < bi.ucntShortest_sum; ++i)
		{
			sprintf(isCommon, "batch:%d\ti:%d\t%d\n", batchID, i, h_isCommon[i]);
			fputs(isCommon, fisCommon);
			fflush(fisCommon);
		}
		free(h_isCommon);
		fclose(fisCommon);
		//debug ends
	}
#endif

	cudppScan(scanplan, d_scan_odata, d_isCommon, bi.ucntShortest_sum);
	CUDA_SAFE_CALL(hipDeviceSynchronize());


#ifdef debug
	//debug; transback h_isCommon and h_scan_odata
	uint32_t *h_scan_odata = (uint32_t *)malloc(baseSize);
	CUDA_SAFE_CALL(hipMemcpy(h_scan_odata, d_scan_odata, baseSize, hipMemcpyDeviceToHost));

	FILE *fResultSum;
	char ResultSum[512];
	sprintf(ResultSum, "ResultSum_a_%u.txt", DOCID_LIMIT / (SM_NUM * THREAD_NUM));
	fResultSum = fopen(ResultSum, "a+");
	
	sprintf(ResultSum, "i:%d\t%u\n", bi.ucntShortest_sum - 1, h_scan_odata[bi.ucntShortest_sum - 1]);
	fputs(ResultSum, fResultSum);
	fflush(fResultSum);
	
	free(h_scan_odata);
	fclose(fResultNum);
	//debug ends
#endif

	saCompact<<<bi.blockNum, THREAD_NUM>>>(d_shortest_lists, d_isCommon, d_scan_odata, d_result, d_queryID_perBlock);
	CUDA_SAFE_CALL(hipDeviceSynchronize());


#ifdef debug
	//debug: transback h_isCommon and h_scan_odata
	uint32_t *h_result = (uint32_t *)malloc(baseSize);
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, baseSize, hipMemcpyDeviceToHost));
	for (int i = 0; i < 600; ++i)
	{
		printf("%u ", h_result[i]);
	}
	printf("\n\n");
	free(h_result);
	//debug ends
#endif


	ucntResult<<<bi.nTotalQueryNum/ 64 + 1, 64>>>(d_scan_odata, d_ucntResult);
	CUDA_SAFE_CALL(hipDeviceSynchronize());


#ifdef debug
	FILE *fResultNum;
	char ResultNum[512];
	sprintf(ResultNum, "ResultNum_a_%u.txt", DOCID_LIMIT / (SM_NUM * THREAD_NUM));
	fResultNum = fopen(ResultNum, "a+");

	uint32_t *ResultNum_temp = (uint32_t*)malloc(bi.nTotalQueryNum * sizeof(uint32_t));
	CUDA_SAFE_CALL(hipMemcpy(ResultNum_temp, d_ucntResult, sizeof(uint32_t) * bi.nTotalQueryNum, hipMemcpyDeviceToHost));
	for (int i = 0; i < bi.nTotalQueryNum; i++)
	{
		sprintf(ResultNum, "%d\t%u\n", i, ResultNum_temp[i]);
		fputs(ResultNum, fResultNum);
		fflush(fResultNum);
	}
#endif
}


void dtohTransfer(const batchInfo &bi) {
	CUDA_SAFE_CALL(hipMemcpy(h_ucntResult, d_ucntResult, sizeof(uint32_t) * bi.nTotalQueryNum, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, sizeof(uint32_t) * h_ucntResult[bi.nTotalQueryNum - 1], hipMemcpyDeviceToHost));
}


void verify(batchInfo bi, checkSum &cs) {
	for (uint32_t i = 0; i < bi.nTotalQueryNum; ++i) {
		if (i) { // all queries except the first one
			if (h_ucntResult[i] - h_ucntResult[i - 1]) {
				uint32_t resultNum = h_ucntResult[i] - h_ucntResult[i - 1];
				cs.checkSum2 += resultNum;
				cs.checkSum3 ^= h_result[h_ucntResult[i - 1]];
				cs.checkSum3 ^= h_result[h_ucntResult[i - 1] + resultNum - 1];
				cs.checkSum3 ^= h_result[h_ucntResult[i - 1] + resultNum / 2];
			}
			else 
				++cs.checkSum1;
		}
		else { // the first query
			if (h_ucntResult[i]) { 
				uint32_t resultNum = h_ucntResult[i];
				cs.checkSum2 += resultNum;
				cs.checkSum3 ^= h_result[0];
				cs.checkSum3 ^= h_result[resultNum - 1];
				cs.checkSum3 ^= h_result[resultNum / 2];
			}
			else 
				++cs.checkSum1;
		}
	}
}


void releaseResource() {
	free(h_result);
	h_result = NULL;

	free(h_ucntResult);
	h_ucntResult = NULL;

	free(h_offset);
	h_offset = NULL;

	free(h_segoffset);
	h_segoffset = NULL;

	free(h_regression_info);
	h_regression_info = NULL;

	free(h_base);
	h_base = NULL;

	free(h_hash_offset);
	h_hash_offset = NULL;

	free(h_hash_info);
	h_hash_info = NULL;

	CUDA_SAFE_CALL(hipHostFree(h_queryID_perBlock));
	h_queryID_perBlock = NULL;


	CUDA_SAFE_CALL(hipFree(d_shortest_lists));
	d_shortest_lists = NULL;

	CUDA_SAFE_CALL(hipFree(d_segoffset));
	d_segoffset = NULL;

	CUDA_SAFE_CALL(hipFree(d_regression_info));
	d_regression_info = NULL;

	CUDA_SAFE_CALL(hipFree(d_base));
	d_base = NULL;

	CUDA_SAFE_CALL(hipFree(d_hash_info));
	d_hash_info = NULL;

	CUDA_SAFE_CALL(hipFree(d_lists));
	d_lists = NULL;

	CUDA_SAFE_CALL(hipFree(d_isCommon));
	d_isCommon = NULL;

	CUDA_SAFE_CALL(hipFree(d_scan_odata));
	d_scan_odata = NULL;

	CUDA_SAFE_CALL(hipFree(d_result));
	d_result = NULL;

	CUDA_SAFE_CALL(hipFree(d_ucntResult));
	d_ucntResult = NULL;

	CUDA_SAFE_CALL(hipFree(d_queryID_perBlock));
	d_queryID_perBlock = NULL;


	CUDPPResult res = cudppDestroyPlan(scanplan);
	if (CUDPP_SUCCESS != res) {
		printf("Error destroying CUDPPPlan\n");
		exit(1);
	}

	cudppDestroy(theCudpp);
}


template <uint64_t beginThreshold, uint64_t endThreshold>
void Run(const string &dataset, unsigned char *ptr_in, unsigned char *ptr_end) {
	ostringstream result;
	result << result_dir << dataset << "_HS" << kExceptedBucketSize << "_SegLRC_Intersection.txt";
	ofstream ofsresult((result.str()).c_str());
	ofsresult << "response: ms/batch" << endl
		      << "throughput: queries/s" << endl
			  << "time: ms" << endl << endl;
	ofsresult << "threshold\tresponse\tthroughput\tcpu\tkernel\ttransfer" << endl;


	batchInfo bi;
	checkSum cs;
	unsigned char *ptr_in_old = ptr_in, *ptr_end_old = ptr_end;

	hipEvent_t start, stop;
	float time_CPU = 0, time_htod = 0, time_kernel = 0, time_dtoh = 0;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	allocateResource();
	
	for (Threshold = beginThreshold; Threshold <= endThreshold; Threshold *= 2) {
		cout << "begin calculating..." << endl;
		if (Threshold < 1024 * 1024) 
			cout << "Threshold: " << Threshold / 1024 << "K";
		else 
			cout << "Threshold: " << Threshold / (1024 * 1024) << "M";

		cout << ", THREAD_NUM: " << THREAD_NUM << endl;

		time_CPU = 0;
		time_htod = 0;
		time_dtoh = 0;
		time_kernel = 0;

		uint32_t nTotalQueryNum = 0;
		for (int iteration = 0; iteration < ITERATION; ++iteration) {
			memset(&cs, 0, sizeof(struct checkSum));

			ptr_in = ptr_in_old;
			ptr_end = ptr_end_old;

			batchID = 0;
			nTotalQueryNum = 0;
			while (ptr_in < ptr_end) {
				hipEventRecord(start, 0);

				bi = CPUPreprocess(ptr_in, ptr_end);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_CPU += elapsedTime;


				hipEventRecord(start, 0);

				htodTransfer(bi);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_htod += elapsedTime;


				hipEventRecord(start, 0);

				kernelInvoke(bi);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_kernel += elapsedTime;


				hipEventRecord(start, 0);

				dtohTransfer(bi);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_dtoh += elapsedTime; 


				verify(bi, cs);


				++batchID;
				nTotalQueryNum += bi.nTotalQueryNum;
			}

		}

		time_CPU /= ITERATION;
		time_htod /= ITERATION;
		time_kernel /= ITERATION;
		time_dtoh /= ITERATION;

		float time_total = time_CPU + time_htod + time_kernel + time_dtoh;

		cout << "number of queries: " << nTotalQueryNum << endl;
		cout << "number of batches: " << batchID << endl;
		cout << "CPU time: " <<  time_CPU << "ms" << endl
			 << "htod time: " <<  time_htod << "ms" << endl
			 << "kernel time: " <<  time_kernel << "ms" << endl
			 << "dtoh time: " <<  time_dtoh << "ms" << endl
			 << "total time: " <<  time_total << "ms" << endl;

		cout << "*********************************************" << endl
			 << "checkSum: " << hex << (cs.checkSum1 ^ cs.checkSum2 ^ cs.checkSum3) << endl
			 << "checkSum1: " << dec << cs.checkSum1 << "\t"
			 << "checkSum2: " << cs.checkSum2 << "\t"
			 << "checkSum3: " << cs.checkSum3 << endl
			 << "*********************************************" << endl << endl;


		float fResponse = time_total / batchID;
		float fThroughput = nTotalQueryNum / time_total * 1000;
		if (Threshold < 1024 * 1024) 
			ofsresult << Threshold / 1024 << "K\t";
		else 
			ofsresult << Threshold / (1024 * 1024) << "M\t";

		ofsresult << fResponse << "\t"
			      << fThroughput << "\t"
			      << time_CPU << "\t"
			      << time_kernel << "\t"
			      << time_htod + time_dtoh << endl;

		sleep(5);
	}

	releaseResource();


	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	ofsresult.close();
}


void terminator(){
	if (ptr != NULL) 
		free(ptr);
	if (patind->m_pind1 != NULL) 
		free(patind->m_pind1);
	if (patind->m_pind2 != NULL) 
		free(patind->m_pind2);
	if (patind != NULL) 
		free(patind);

	ptr = NULL;
	patind->m_pind1 = NULL;
	patind->m_pind2 = NULL;
	patind = NULL;
}


void runTest(int argc, char **argv) {
	if (argc < 2) {
		cout << "wrong number of arguments" << endl;
		exit(1);
	}
	string dataset = argv[1];

	if (argc > 2) {
		kExceptedBucketSize = atoi(argv[2]);
	}
	cout << "HS" << kExceptedBucketSize << "_SegLRC_Intersection" << endl;
	cout << "dataset = " << dataset << endl;

	if (argc > 3) {
		CUDA_SAFE_CALL(hipSetDevice(strtol(argv[3], NULL, 10)));
	}
	else {
		CUDA_SAFE_CALL(hipSetDevice(0));
	}


	string queryset = dataset_dir + dataset + "/" + dataset + ".query";

	string offset_file = data_dir + dataset + ".segNum";
	string segoffset_file = data_dir + dataset + ".segOffset";

	string regression_info_file = regressioninfo_dir + dataset + ".regression_info";
	string base_file = regressioninfo_dir + dataset + ".ind2_distance_base";
	string hash_offset_file = regressioninfo_dir + dataset + ".HashSeg_offset";
	string hash_info_file = regressioninfo_dir + dataset + ".HashSeg_info";

	struct stat buf;

	FILE *foffset = fopen(offset_file.c_str(), "rb");
	if (!foffset) {
		cout << offset_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(offset_file.c_str(), &buf);
	h_offset = (uint32_t *)malloc(buf.st_size);			
	readFile((unsigned char *)h_offset, foffset);
	fclose(foffset);

	
	FILE *fsegoffset = fopen(segoffset_file.c_str(), "rb");
	if (!fsegoffset) {
		cout << segoffset_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(segoffset_file.c_str(), &buf);
	segoffset_size = buf.st_size;							
	h_segoffset = (uint32_t *)malloc(segoffset_size);	
	readFile((unsigned char *)h_segoffset, fsegoffset);
	fclose(fsegoffset);


	FILE *fregression_info = fopen(regression_info_file.c_str(), "rb");
	if (!fregression_info) {
		cout << regression_info_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(regression_info_file.c_str(), &buf);
	regression_info_size = buf.st_size;
	h_regression_info = (regression_info_t*)malloc(regression_info_size);
	readFile((unsigned char *)h_regression_info, fregression_info);
	fclose(fregression_info);

	FILE *fbase = fopen(base_file.c_str(), "rb");
	if (!fbase) {
		cout << base_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(base_file.c_str(), &buf);
	base_size = buf.st_size;
	h_base = (int *)malloc(base_size);
	readFile((unsigned char *)h_base, fbase);
	fclose(fbase);


	FILE *fhash_offset = fopen(hash_offset_file.c_str(), "rb");
	if (!fhash_offset) {
		cout << hash_offset_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(hash_offset_file.c_str(), &buf);
	hash_offset_size = buf.st_size;
	h_hash_offset = (uint32_t*)malloc(hash_offset_size);
	readFile((unsigned char*)h_hash_offset, fhash_offset);
	fclose(fhash_offset);

	FILE *fhash_info = fopen(hash_info_file.c_str(), "rb");
	if (!fhash_info) {
		cout << hash_info_file << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(hash_info_file.c_str(), &buf);
	hash_info_size = buf.st_size;
	h_hash_info = (uint32_t*)malloc(hash_info_size);
	readFile((unsigned char*)h_hash_info, fhash_info);
	fclose(fhash_info);

	
	FILE *fquery = fopen(queryset.c_str(),"rb");
	if (!fquery) {
		cout << queryset << " open failed\terr code: " << errno << endl;
		exit(1);
	}
	stat(queryset.c_str(), &buf);  
	uint32_t querysize = buf.st_size;
	ptr = (unsigned char *)malloc(querysize);					
	readFile(ptr, fquery);
	fclose(fquery);


	as_load_atind(data_dir.c_str(), dataset.c_str());
	
	// <1K, 2M>
	Run<1024, 2 * 1024 * 1024>(dataset, ptr, ptr + querysize);

	terminator();
}


int main(int argc, char **argv) {
    runTest(argc, argv);

	return 0;
}
