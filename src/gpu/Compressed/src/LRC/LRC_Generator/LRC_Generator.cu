#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include </usr/include/sys/stat.h>
#include <errno.h>

#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>

#include "LRC_Generator_kernel.cu"

using namespace std;


string dataset_dir = "/media/indexDisk/naiyong/dataset/";
string regressioninfo_dir = "/media/indexDisk/naiyong/data/LRC/Generator/";
string result_dir = "/media/indexDisk/naiyong/result/LRC/Generator/";

ofstream ofsresult;
const uint32_t interval = 100 * 1000;
uint32_t intervalNum = MAX_LIST_LEN / interval;


at_search_ind_t *patind = NULL;



inline uint64_t div_roundup(uint64_t v, uint32_t divisor) {
	    return (v + (divisor - 1)) / divisor;
}

inline uint32_t gccbits(const uint32_t v) {
	return v == 0 ? 0 : 32 - __builtin_clz(v);
}

inline uint32_t maxbits(const uint32_t *in, uint32_t nvalue) {
	uint32_t accumulator = 0;
	for (uint32_t i = 0; i < nvalue; ++i) {
		accumulator |= in[i];
	}
	return gccbits(accumulator);
}


#define BUFFER_SIZE 2048
unsigned char buffer[BUFFER_SIZE];
inline void readFile(unsigned char* ptr, FILE* fp){
	uint64_t count = 0, bytes;
	while ((bytes = fread(ptr+count, 1, BUFFER_SIZE, fp))>0) {
		count += bytes;
	}
}

void as_load_atind(const char *dbi_dir, const char *ind_name) {
	patind = (at_search_ind_t *)malloc(sizeof (*patind));
	struct stat buf;

	char file_name[MAX_PATH_LEN];
	patind->fd_ind1 = 0;
	patind->fd_ind2 = 0;
	patind->m_pind1 = 0;
	patind->m_pind2 = 0;

	// ind1
	sprintf (file_name, "%s%s.ind1", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind1 = buf.st_size;
	patind->m_tcount = buf.st_size / sizeof (at_term_ind1_t);
	patind->fd_ind1 = fopen(file_name, "rb");
	patind->m_pind1 = (at_term_ind1_t *)malloc(buf.st_size);
	readFile((unsigned char*)patind->m_pind1 , patind->fd_ind1);

	//ind2
	sprintf (file_name, "%s%s.ind2", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind2 = buf.st_size;
	patind->fd_ind2 = fopen(file_name,"rb");
	patind->m_pind2 = (unsigned char*) malloc(buf.st_size);
	readFile(patind->m_pind2 , patind->fd_ind2);
}



void generateInd2Distance(const string &dataset) {
	struct stat buf;

	// read at_regression_info file
	string strRegressionInfo = regressioninfo_dir + dataset + ".regression_info";
	FILE *fRegressionInfo = fopen(strRegressionInfo.c_str(), "rb+");
	if (!fRegressionInfo) {
		cout << "read binary file " << strRegressionInfo << " failed; error: " << endl;
		exit(1);
	}

	stat(strRegressionInfo.c_str(), &buf);
	uint32_t szRegressionInfo = buf.st_size;
	regression_info_t *h_regressioninfo = (regression_info_t *)malloc(szRegressionInfo);
	if (h_regressioninfo == 0) {
		perror("alloc error\n");
		exit(1);
	}
	readFile((unsigned char *)h_regressioninfo, fRegressionInfo);
	// read ends


	// create ind2_distance
	string strDistance = regressioninfo_dir + dataset + ".ind2_distance";
	FILE *fDistance = fopen(strDistance.c_str(), "wb+");
	if (!fDistance) {
		cout << strDistance << " create failed" << endl;
		exit(1);
	}

	uint64_t ind2Size = patind->sz_ind2;
	uint32_t *VDs = (uint32_t *)malloc(ind2Size);
	if (VDs == 0) {
		perror("VDs error\n");
		exit(1);
	}
	// create ends


	// cuda allocation & transfer
	CUDA_SAFE_CALL(hipMalloc((void**)&d_list,  MAX_LIST_LEN * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_distance, MAX_LIST_LEN * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_regressioninfo, szRegressionInfo));

	CUDA_SAFE_CALL(hipMemcpy(d_regressioninfo, h_regressioninfo, szRegressionInfo, hipMemcpyHostToDevice));


	vector<uint64_t> nBitsNeeded(intervalNum, 0);
	vector<uint64_t> length(intervalNum, 0);

	uint32_t nListNum = patind->sz_ind1 / sizeof(at_term_ind1_t);
	for (uint32_t i = 0; i < nListNum; ++i) {
		// progress
		if (!(i % (nListNum / 10))) {
			printf("%f%%...\n", ((float)i) / nListNum * 100);
		}

	    at_term_ind1_t *pind1 = patind->m_pind1 + i;
		uint32_t nListLen = pind1->m_urlcount;
		uint32_t *h_list = (uint32_t *)(patind->m_pind2 + pind1->m_off);
		int *h_distance = (int *)(VDs + pind1->m_off / sizeof(uint32_t));


		if (nListLen > 1) {
			CUDA_SAFE_CALL(hipMemcpy(d_list, h_list, nListLen * sizeof(uint32_t), hipMemcpyHostToDevice));

			LRC_Generator<<<nListLen / THREAD_NUM + 1, THREAD_NUM>>>(d_list, d_regressioninfo, d_distance, nListLen, i);
			CUDA_SAFE_CALL(hipDeviceSynchronize());

			CUDA_SAFE_CALL(hipMemcpy(h_distance, d_distance, nListLen * sizeof(int), hipMemcpyDeviceToHost));


			// select the smallest one in the h_distance
			int nBase = h_distance[0];
			for (uint32_t docIndex = 1; docIndex < nListLen; ++docIndex) {
				if (h_distance[docIndex] < nBase) 
					nBase = h_distance[docIndex];
			}

			// reduction
			for (uint32_t docIndex = 0; docIndex < nListLen; ++docIndex) {
				h_distance[docIndex] -= nBase;
			}

			h_regressioninfo[i].iIntercept += nBase;
		}
		else 
			h_distance[0] = 0;


		uint32_t idx = nListLen / interval;
		length[idx] += nListLen;
		uint32_t mb = maxbits(reinterpret_cast<const uint32_t *>(h_distance), nListLen);
		uint32_t nwords = div_roundup(nListLen * mb, 32); 
		nBitsNeeded[idx] += nwords * 32;
	}

	fseek(fRegressionInfo, 0, SEEK_SET);
	fwrite(h_regressioninfo, szRegressionInfo, 1, fRegressionInfo);

	fwrite(VDs, ind2Size, 1, fDistance);
	fflush(fDistance);


	ofsresult << "length\t" << "Compression (bits/interger)" << endl;
	for (uint32_t i = 0; i < intervalNum; ++i) {
		if (length[i] > 0) {
			nBitsNeeded[i] /= length[i];
			ofsresult << "[" << (i * interval) / 1000 << "K, "
				      << ((i+1) * interval) / 1000 << "K)\t"
					  << nBitsNeeded[i] << endl;
		}
	}


	free(h_regressioninfo);
	free(VDs);

	CUDA_SAFE_CALL(hipFree(d_list));
	CUDA_SAFE_CALL(hipFree(d_distance));
	CUDA_SAFE_CALL(hipFree(d_regressioninfo));

	fclose(fRegressionInfo);
	fclose(fDistance);
}


// average edition
void getRegressionInfo(const uint32_t *pList, uint32_t nLen, regression_info_t &regressionInfo, stat_info_t &statInfo) {
	if (nLen == 0) {
		regressionInfo.fSlope = 1;
		regressionInfo.iIntercept = 0;
		regressionInfo.fSearchRange = 0;

		statInfo.dRSquare = 1;
		statInfo.dContractionRatio = 0;

		return;
	}

	if (nLen == 1) {
		regressionInfo.fSlope = 1;
		regressionInfo.iIntercept = pList[0];
		regressionInfo.fSearchRange = 0;

		statInfo.dRSquare = 1;
		statInfo.dContractionRatio = 0;

		return;
	}


	// average
	double dXA = 0, dYA = 0;
	for (uint32_t i = 0; i < nLen; ++i) {
		dXA += i;
		dYA += pList[i];
	}
	dXA /= nLen;
	dYA /= nLen;

	// diff sum
	double dDiffSumX = 0, dDiffSumXY = 0, dDiffSumY = 0;
	for (uint32_t i = 0; i < nLen; ++i) {
		double dValueY = (double)(pList[i]);
		dDiffSumX += (double)((i - dXA) * (i - dXA));
		dDiffSumXY += (double)((i - dXA) * (dValueY - dYA));
		dDiffSumY += (double)((dValueY - dYA) * (dValueY - dYA));
	}

	// slope and intercept
	double dSlope = dDiffSumXY / dDiffSumX;
	double dIntercept = dYA - dSlope * dXA;

	// furthest points
	double dRangeLeft = 0, dRangeRight = 0;
	for (uint32_t i = 0; i < nLen; ++i) {
		double dPrivateX = (pList[i] - dIntercept) / dSlope;
		if (dPrivateX - i > dRangeLeft) {
			dRangeLeft = dPrivateX - i;
		}
		else if (dPrivateX - i < dRangeRight) {
			dRangeRight = dPrivateX - i;
		}
	}

	//regression info
	regressionInfo.fSlope = float(dSlope);
	regressionInfo.iIntercept = int(dIntercept);
	regressionInfo.fSearchRange = float(dRangeLeft - dRangeRight);

	// stat info
	statInfo.dRSquare = (dDiffSumXY * dDiffSumXY) / (dDiffSumX * dDiffSumY);
	statInfo.dContractionRatio = (dRangeLeft - dRangeRight) / nLen;
}


// generate detailed regression data
void RunGetRegressionInfo(const string &dataset) {
	// open file
	string strRegressionInfo = regressioninfo_dir + dataset + ".regression_info";
	FILE *fRegressionInfo = fopen(strRegressionInfo.c_str(), "wb+");
	if (!fRegressionInfo) {
		cout << "open binary file " << strRegressionInfo << " failed; error:" << endl;
		exit(1);
	}


	vector<stat_info_t> statInfo(intervalNum);
	memset(&statInfo[0], 0, intervalNum * sizeof(stat_info_t));
	vector<uint32_t> count(intervalNum, 0);

	uint32_t nListNum = patind->sz_ind1 / sizeof(at_term_ind1_t);
	vector<regression_info_t> regressionInfo(nListNum);

	for (uint32_t i = 0; i < nListNum; ++i) {
		at_term_ind1_t *pind1 = patind->m_pind1 + i;
		uint32_t nListLen = pind1->m_urlcount;
		uint32_t *pList = (uint32_t *)(patind->m_pind2 + pind1->m_off);

		stat_info_t thisStatInfo;
		memset(&thisStatInfo, 0, sizeof(stat_info_t));
		getRegressionInfo(pList, nListLen, regressionInfo[i], thisStatInfo);

		if (nListLen > 1) {
			uint32_t idx = nListLen / interval;
			++count[idx];
			statInfo[idx].dRSquare += thisStatInfo.dRSquare;
			statInfo[idx].dContractionRatio += thisStatInfo.dContractionRatio;
		}
	}
	// write result to binary file
	fwrite(&regressionInfo[0], sizeof(regression_info_t), nListNum, fRegressionInfo);
	fflush(fRegressionInfo);


	ofsresult << "length\t" << "count\t"
			  << "R^2\t" << "Ratio" << endl;
	for (uint32_t i = 0; i < intervalNum; ++i) {
		if (count[i] > 0) {
			statInfo[i].dRSquare /= count[i];
			statInfo[i].dContractionRatio /= count[i];
			ofsresult << "[" << (i * interval) / 1000 << "K, "
				      << ((i+1) * interval) / 1000 << "K)\t"
					  << count[i] << "\t" 
					  << statInfo[i].dRSquare << "\t"
					  << statInfo[i].dContractionRatio << endl;
		}
	}
	ofsresult << endl;


	fclose(fRegressionInfo);
}


// Do some free operations
void terminator(){
	if (patind->m_pind1 != NULL) 
		free(patind->m_pind1);
	if (patind->m_pind2 != NULL) 
		free(patind->m_pind2);
	if (patind != NULL) 
		free(patind);

	patind->m_pind1 = NULL;
	patind->m_pind2 = NULL;
	patind = NULL;
}

void runTest(int argc, char** argv) {
	if (argc < 2) {
		std::cout << "wrong number of arguments" << std::endl;
		exit(1);
	}
	string dataset = argv[1];
	cout << "LRC_Generator" << endl;
	cout << "dataset = " << dataset << endl;

	if (argc > 2) {
		CUDA_SAFE_CALL(hipSetDevice(strtol(argv[2], NULL, 10)));
	}
	else // default set to Tesla
	{
		CUDA_SAFE_CALL(hipSetDevice(0));
	}

	string index_dir = dataset_dir + dataset + "/";
	as_load_atind(index_dir.c_str(), dataset.c_str());


	string result = result_dir + dataset + "_LRC_Generator.txt";
	ofsresult.open(result.c_str());

	cout << "Getting regression info..." << endl;
	RunGetRegressionInfo(dataset);

	cout << "Generating ind2 distance..." << endl;
	generateInd2Distance(dataset);

	ofsresult.close();


	terminator();
}


int main(int argc, char **argv) {
    runTest(argc, argv);

	return 0;
}

