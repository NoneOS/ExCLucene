#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdint.h>


#define CUDA_SAFE_CALL(call) {                                         \
    hipError_t err = call;                                              \
    if (hipSuccess != err) {                                          \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",  \
                __FILE__, __LINE__, hipGetErrorString( err) );        \
        exit(EXIT_FAILURE);                                            \
    }                                                                  \
}


#define	MAX_PATH_LEN   1024
#define THREAD_NUM     256
#define MAX_LIST_LEN   30000000

// ind1
typedef struct _at_term_ind1 {
	uint32_t m_urlcount;  // length 
	uint64_t m_off;       // offset (in bytes)
} at_term_ind1_t;

typedef struct _at_search_ind {
	FILE* fd_ind1;				
	FILE* fd_ind2;			
	uint64_t sz_ind1;	
	uint64_t sz_ind2;
	at_term_ind1_t *m_pind1;
	unsigned char *m_pind2;	
	uint32_t m_tcount;		
} at_search_ind_t;

typedef struct _regression_info {
	float fSlope;
	float fIntercept;
	uint32_t nRangeLeft;
	uint32_t nRangeRight;
} regression_info_t;

typedef struct _stat_info {
	double dRSquare;
	double dContractionRatio;
	uint32_t nBitsNeeded;
} stat_info_t;


uint32_t *d_list;
regression_info_t *d_regressioninfo;
int *d_distance;



__device__ float DcalcVDis(uint32_t eleIndex, uint32_t nDocID, float fSlope, float fIntercept) {
	return (nDocID - (eleIndex * fSlope + fIntercept));
}


__global__ void SegLRC_Generator(uint32_t *d_list, regression_info_t *d_regressioninfo, int *d_distance, uint32_t offset, uint32_t nListLen) {
	uint32_t tid = threadIdx.x;
	uint32_t bid = blockIdx.x;
	uint32_t eleIndex = bid * blockDim.x + tid;

	float fSlope;
	float fIntercept;
	uint32_t nDocID;
	float fOriGap;
	int nDistance;
	float fYTheory;
	int nY;

	if (eleIndex < nListLen) {
		fSlope = (d_regressioninfo + offset + bid)->fSlope;
		fIntercept = (d_regressioninfo + offset + bid)->fIntercept;
		nDocID = d_list[eleIndex];
		fOriGap = DcalcVDis(tid, nDocID, fSlope, fIntercept);
		nDistance = (int)fOriGap;
		
		fYTheory = tid * 1 * fSlope + fIntercept;
		nY = (int)fYTheory;

		int nDistanceMinus = nDistance, nDistancePlus = nDistance;
		while (1) {
			if ((uint32_t)(nY + nDistanceMinus) == nDocID) {
				d_distance[eleIndex] = nDistanceMinus;
				break;
			}

			if ((uint32_t)(nY + nDistancePlus) == nDocID) {
				d_distance[eleIndex] = nDistancePlus;
				break;
			}

			--nDistanceMinus;
			++nDistancePlus;
		}
	}
	__syncthreads();

}

#endif // #ifndef _TEMPLATE_KERNEL_H_


