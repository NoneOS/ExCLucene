#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include </usr/include/sys/stat.h>
#include </usr/include/sys/time.h>
#include <errno.h>

#include <iostream>
#include <string>
#include <vector>

#include <cudpp.h>
#include <hip/hip_runtime.h>

using namespace std;


#include "cuda_binarySearch_kernel.cu"

#define ITERATION 2 
#define fullProcess
//#define displayCPU
//#define displayKernel


unsigned char* ptr;
at_search_ind_t *patind;

#define BUFFER_SIZE 2048
unsigned char buffer[BUFFER_SIZE];
unsigned int GetTickCount()
{
	struct timeval tv;
	if (!gettimeofday(&tv, NULL))  //gettimeofday, ?ɹ???0
	{
		return tv.tv_sec * 1000 + tv.tv_usec / 1000;
	}
	else
	{
		return 0;
	}
}


inline void readFile(unsigned char* ptr, FILE* fp){
	int start = GetTickCount();
	uint64_t count = 0, bytes;
	while ((bytes = fread(ptr+count, 1, BUFFER_SIZE, fp))>0) {
		count += bytes;
	}

	cout << "size: " << count / double(1024 * 1024) << "MB" << endl;
	cout << "time: " << GetTickCount() - start << "ms" << endl;
}

at_search_ind_t* as_load_atind(const char *dbi_dir, const char *ind_name)
{
	
	patind = (at_search_ind_t *)malloc(sizeof (*patind));
	struct stat buf;

	char file_name[MAX_PATH_LEN];
	patind->fd_ind1 = 0;
	patind->fd_ind2 = 0;
	patind->m_pind1 = 0;
	patind->m_pind2 = 0;

	// ind1
	sprintf (file_name, "%s%s.ind1", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind1 = buf.st_size;
	patind->m_tcount = buf.st_size / sizeof (at_term_ind1_t);
	patind->fd_ind1 = fopen(file_name, "rb");
	patind->m_pind1 = (at_term_ind1_t *)malloc(buf.st_size);
	readFile((unsigned char*)patind->m_pind1 , patind->fd_ind1);

	//ind2
	sprintf (file_name, "%s%s.ind2", dbi_dir, ind_name);
	cout << "reading " << file_name << endl;
	stat(file_name, &buf);
	patind->sz_ind2 = buf.st_size;
	patind->fd_ind2 = fopen(file_name,"rb");
	patind->m_pind2 = (unsigned char*) malloc(buf.st_size);
	readFile(patind->m_pind2 , patind->fd_ind2);

	return patind;
}



int last;
int* result_seq;
int gggcount  = 0;


struct single_keyword_struct
{
	unsigned int queryID;  //??0??ʼ
	unsigned int length;	//Ͱ??
	unsigned int offset;	//?ڵ????????е?ƫ??
};

char resultFileName[100];
FILE *fpTotalStat;


//Ԥ??????Դ???????Ĵ???
//??ȷ??0,???�1??????ģʽ??
int allocateResource(/*host??Դ*/unsigned int **h_constant, uint16_t **h_queryID_perBlock, unsigned int **h_result, unsigned int **h_ucntResult, unsigned char **h_lists, unsigned char **h_bloom, regresstion_info_t **h_pSLinearInfoDetail, float **h_fLinearInfo, /*device??Դ*/uint16_t **d_queryID_perBlock, unsigned int **d_lists, unsigned int **d_isCommon, unsigned int **d_scan_odata, unsigned int **d_result, unsigned int **d_ucntResult, unsigned int size_d_lists, unsigned char **d_bloom, unsigned int **d_batchInfo, float ** d_fLinearInfo, /*cudpp??Դ*/CUDPPConfiguration *config)
{
	// by ysharp: initialize cudpp library
	cudppCreate(&theCudpp);



	//host??Դ
	*h_result = (unsigned int*)malloc(baseSize);
	*h_ucntResult = (unsigned int*)malloc(50000 * sizeof(unsigned int));
	CUDA_SAFE_CALL(hipHostMalloc((void**)h_queryID_perBlock, 65535 * sizeof(uint16_t)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)h_constant, 163800 * sizeof(unsigned int)));
	//CUDA_SAFE_CALL(hipHostMalloc((void**)h_fLinearInfo, 16380 * sizeof(float)));

	//device??Դ
	CUDA_SAFE_CALL(hipMalloc((void**)d_isCommon, baseSize));
	CUDA_SAFE_CALL(hipMalloc((void**)d_scan_odata, baseSize));
	CUDA_SAFE_CALL(hipMalloc((void**)d_result, baseSize));
	CUDA_SAFE_CALL(hipMalloc((void**)d_ucntResult, 50000 * sizeof(unsigned int)));	//no more than 5000 queries in a batch, else the response time will be unacceptable
	CUDA_SAFE_CALL(hipMalloc((void**)d_lists, size_d_lists));
	CUDA_SAFE_CALL(hipMalloc((void**)d_queryID_perBlock, 65535 * sizeof(uint16_t)));
	CUDA_SAFE_CALL(hipMalloc((void**)d_batchInfo, 1024 * 1024));
//	CUDA_SAFE_CALL(hipMalloc((void**)d_fLinearInfo, 163800 * sizeof(float)));

	/*
	//bloomfilter
	FILE *fpBloom = fopen("data/gov/bloom5.ind", "rb");
	if (!fpBloom)
	{
		perror("can't open bloom.ind");
		exit(1);
	}
	unsigned int nSize = 0;
	struct stat buf;
	stat("data/gov/bloom5.ind", &buf); 
	nSize = buf.st_size;
	printf("size of bloom.ind:%u", nSize);
	*h_bloom = (unsigned char *)malloc(nSize);
	if (!(*h_bloom))
	{
		perror("h_bloom allocated failed");
	}
	printf("reading bloom.ind...\n");
	readFile(*h_bloom, fpBloom);
	fclose(fpBloom);
	*/

	/*
	//at_regression_info
	FILE *fpLinear = fopen("data/gov/at_regression_info", "rb");
	if (!fpLinear)
	{
		perror("can't open at_regression_info");
		exit(1);
	}
	unsigned int nSize = 0;
	struct stat buf;
	stat("data/gov/at_regression_info", &buf);
	nSize = buf.st_size;
	*h_pSLinearInfoDetail = (regresstion_info_t *)malloc(nSize);
	if (!(*h_pSLinearInfoDetail))
	{
		perror("h_pSLinearInfoDetail allocated failed");
		exit(1);
	}
	printf("reading at_regression_info\n");
	readFile((unsigned char *)(*h_pSLinearInfoDetail), fpLinear);
	fclose(fpLinear);
	fpLinear = NULL;
	// at_regression_info ends
	*/

	//CUDA_SAFE_CALL(hipMalloc((void**)d_bloom, nSize));
	//CUDA_SAFE_CALL(hipMemcpy(*d_bloom, *h_bloom, nSize, hipMemcpyHostToDevice));

	if (!(*h_result && *d_lists && *d_isCommon && *d_scan_odata && *d_result))
	{
		return 1;
	}

	//transfer lists
	printf("transferring ind2...\n");
	CUDA_SAFE_CALL(hipMemcpy(*d_lists, *h_lists, size_d_lists, hipMemcpyHostToDevice));

	//cudpp alloc
	(*config).op = CUDPP_ADD;
	(*config).datatype = CUDPP_INT;
	(*config).algorithm = CUDPP_SCAN;
	(*config).options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
	CUDPPResult result = cudppPlan(theCudpp, &scanplan, *config, 30000000, 1, 0);

	if (CUDPP_SUCCESS != result)
	{
		return 1;
	}

	return 0;
}

unsigned int nBFQueryNum = 0;
FILE *fpCPUPre = NULL;
batchInfo CPUPreprocess(/*host resource*/unsigned int *h_constant, unsigned int *h_queries, unsigned int *h_startBlockId, unsigned int *h_queriesOffset, uint16_t *h_queryID_perBlock, regresstion_info_t *h_pSLinearInfoDetail, float *h_fLinearInfo, /*query resource*/at_search_ind_t *patind, char **ptr_in, char **ptr_end, vector<singleKeyword_t> *pvSSingleKeyword)
{
	//CPU preprocess temp memory
	query_input_t input;  //save current query
	testCell tc;			//save detailed info of the current query
	at_term_ind1_t *pind1_temp;		//temp ind1 pointer for getting detailed query info
	register unsigned int i, j, k;	//loop control variables
	register unsigned int tcount = 0;	//number of terms; used frequently
	batchInfo bi;	//infor of this batch
	unsigned int current_blockNum;	//block number for current query
	pvSSingleKeyword->clear();
	//CPU preprocess temp memory ends

		
	//local temp variables
	unsigned int ucntShortest_sum = 0; 
	unsigned int blockNum= 0;
	unsigned int query_num = 0;
	unsigned int nTotalQueryNum= 0;  //queries  whose lists number is more than 1
	unsigned int queries_offset = 0;	//offset for h_queries
	singleKeyword_t SSingleKey;  //struct of singleKeyword
	unsigned int nRoute = ROUTE_BINARY;
	//local temp variables end

	while (ucntShortest_sum <= DOCID_LIMIT && *ptr_in < *ptr_end)
	{
		//get current query
		input.tnum = **ptr_in;
		*ptr_in += sizeof(unsigned int);
		memcpy(input.tno, *ptr_in, sizeof(unsigned int) * input.tnum);
		*ptr_in += sizeof(unsigned int) * input.tnum;

		//get detailed query info: ucnt and uwlist
		tc.tcount = input.tnum;
		tcount = tc.tcount;

#ifdef debug
		printf("tcount:%u\n", tcount);
#endif

		for (i = 0; i < tcount; ++i)
		{
			pind1_temp = patind->m_pind1 + input.tno[i];
			tc.ucnt[i] = pind1_temp->m_urlcount;  
			tc.uwlist[i] = (pind1_temp->m_off) / sizeof(unsigned int);	
			tc.tno[i] = input.tno[i];
			
#ifdef debug
			printf("tno:%d\t", input.tno[i]);
			printf("ucnt:%u\t", tc.ucnt[i]);
			printf("offset:%u\n", tc.uwlist[i]);
#endif
		}

		//single keyword process
		if (1 == tcount)
		/*if (2 != tcount)*/
		{
			SSingleKey.nLen = tc.ucnt[0];
			SSingleKey.nOffset = tc.uwlist[0];
			SSingleKey.nQueryIDInBatch = nTotalQueryNum;
			pvSSingleKeyword->push_back(SSingleKey);
			nTotalQueryNum++;
			continue;
		}

		//insertion sort
		for (i = 1; i < tcount; ++i)
		{
			k = i;
			unsigned int uwlist_tmp = tc.uwlist[i];
			unsigned int ucnt_tmp = tc.ucnt[i];
			unsigned int tno_tmp = tc.tno[i];

			while (k && ucnt_tmp < tc.ucnt[k - 1])
			{
				tc.uwlist[k] = tc.uwlist[k - 1];
				tc.ucnt[k] = tc.ucnt[k - 1];
				tc.tno[k] = tc.tno[k - 1];
				k--;
			};

			if (k != i)
			{
				tc.uwlist[k] = uwlist_tmp;
				tc.ucnt[k] = ucnt_tmp;
				tc.tno[k] = tno_tmp;
			}
		}

#ifdef displayCPU
		printf("tcount: %u\n", tcount);
		for (int m = 0; m < tcount; ++m)
		{
			printf("ucnt:%u\n", tc.ucnt[m]);
		}
#endif

		//calculate block number needed by current query
		current_blockNum = tc.ucnt[0] % THREAD_NUM ? tc.ucnt[0] / THREAD_NUM + 1 : tc.ucnt[0] / THREAD_NUM;

		//set host memory for constant memory
		h_startBlockId[query_num] = blockNum;
		h_baseOffset[query_num] = ucntShortest_sum;
		h_queriesOffset[query_num] = queries_offset;
		//process ends

		//copy the query from tc to h_queries
		*(h_queries + queries_offset) = tc.tcount;
		*(h_queries + queries_offset + 1) = ucntShortest_sum;
		*(h_queries + queries_offset + 2) = 0;//linearInfo_offset;
		*(h_queries + queries_offset + 3) = nRoute;
		memcpy(h_queries + queries_offset + 4, tc.ucnt, sizeof(unsigned int) * tc.tcount);
		memcpy(h_queries + queries_offset + 4 + tc.tcount, tc.uwlist, sizeof(unsigned int) * tc.tcount);
		//copy ends

		//set queryID for each block
		for (k = 0; k < current_blockNum; ++k)
		{
			h_queryID_perBlock[blockNum + k] = query_num;	
		}

		//set several local variables for next loop
		blockNum += current_blockNum;
		ucntShortest_sum += tc.ucnt[0];
		queries_offset += tc.tcount * 2 + 4;
		query_num++;
		nTotalQueryNum++;
	};

	//whether the base arrays is not big enough
	if (baseSize / sizeof(unsigned int) < ucntShortest_sum)
	{
		printf("!!!!!!!!!!!!!!!!!!!!!!!!\nresource allocated as d_isCommon is insuffient\n!!!!!!!!!!!!!!!!!!!!!!!!\n");
		exit(1);
	}


	//whether the blockNum is over 65535
	if (65535 < blockNum)
	{
		printf("!!!!!!!!!!!!!!!!!!!!!!!!\nblockNum is over 65535\n!!!!!!!!!!!!!!!!!!!!!!!!\n");
		exit(1);
	}

	//prepare for return value
	bi.blockNum = blockNum;
	bi.queryNum = query_num; 
	bi.constantUsedInByte = (query_num * 3 + queries_offset + batchInfoElementNum) * sizeof(unsigned int);  
	bi.ucntShortest_sum = ucntShortest_sum;
	bi.nTotalQueryNum = nTotalQueryNum;


	//printf("constant used: %u\n", bi.constantUsedInByte);

	//integrate five arrays into h_constant
	memcpy(h_constant, &bi, sizeof(struct batchInfo));
	memcpy(h_constant + batchInfoElementNum, h_startBlockId, sizeof(unsigned int) * query_num);
	//memcpy(h_constant + batchInfoElementNum + query_num, h_baseOffset, sizeof(unsigned int) * query_num);
	memcpy(h_constant + batchInfoElementNum + query_num, h_queriesOffset, sizeof(unsigned int) * query_num);
	memcpy(h_constant + batchInfoElementNum + query_num * 2, h_queries, queries_offset * sizeof(unsigned int));


	return bi;
}

void htodTransfer(unsigned int *h_constant, uint16_t *h_queryID_perBlock, float *h_fLinearInfo, unsigned int *d_constantLocal, uint16_t *d_queryID_perBlock, batchInfo bi, unsigned int *d_batchInfo, float *d_fLinearInfo)
{
//	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_constant), h_constant, bi.constantUsedInByte));
	CUDA_SAFE_CALL(hipMemcpy(d_batchInfo, h_constant, bi.constantUsedInByte, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_queryID_perBlock, h_queryID_perBlock, bi.blockNum * sizeof(uint16_t), hipMemcpyHostToDevice));

}

void kernelInvoke(/*host resource*/batchInfo bi,  /*device resource*/unsigned int *d_lists, unsigned int *d_isCommon, unsigned int *d_scan_odata, unsigned int *d_result, unsigned int *d_ucntResult, uint16_t *d_queryID_perBlock, unsigned int *d_batchInfo, float *d_fLinearInfo)
{
	
#ifdef debug	
	printf("blockNum:%u\n",bi.blockNum);
#endif

	mqSearch<<<bi.blockNum, THREAD_NUM >>>(d_batchInfo, d_lists, d_isCommon, d_queryID_perBlock);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

#ifdef displayKernel
	if (bi.ucntShortest_sum == 884)
	{
	//debug; transback h_isCommon and h_scan_odata
		FILE *fileTemp = fopen("new_temp_file", "w+");
		char tempString[256];
	unsigned int *h_isCommon = (unsigned int *)malloc(baseSize);
	CUDA_SAFE_CALL(hipMemcpy(h_isCommon, d_isCommon, baseSize, hipMemcpyDeviceToHost));
	for (int i = 0; i < bi.ucntShortest_sum; ++i)
	{
		printf("%d ", h_isCommon[i]);
		sprintf(tempString, "%u:%u\n", i, h_isCommon[i]);
		fputs(tempString, fileTemp);
		fflush(fileTemp);
	}
	printf("\n\n");
	free(h_isCommon);
		fclose(fileTemp);
	//debug ends
	}
#endif

	cudppScan(scanplan, d_scan_odata, d_isCommon, bi.ucntShortest_sum);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifdef displayKernel
	//debug; transback h_isCommon and h_scan_odata
	unsigned int *h_scan_odata = (unsigned int *)malloc(baseSize);
	CUDA_SAFE_CALL(hipMemcpy(h_scan_odata, d_scan_odata, baseSize, hipMemcpyDeviceToHost));
	//for (int i = 0; i < 600; ++i)
	if (bi.ucntShortest_sum == 884)
	{
		printf("resNumFromScan: %u ", h_scan_odata[883]);
	}
	printf("\n\n");
	free(h_scan_odata);
	//debug ends
#endif

#ifdef fullProcess
	
	saCompact<<<bi.blockNum, THREAD_NUM>>>(d_batchInfo, d_lists, d_isCommon, d_scan_odata, d_result, d_queryID_perBlock);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	

#ifdef debug
	//debug; transback h_isCommon and h_scan_odata
	unsigned int *h_result = (unsigned int *)malloc(baseSize);
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, baseSize, hipMemcpyDeviceToHost));
	for (int i = 0; i < 600; ++i)
	{
		printf("%u ", h_result[i]);
	}
	printf("\n\n");
	free(h_isCommon);
	//debug ends
#endif

	ucntResult<<<bi.queryNum / 64+ 1, 64>>>(d_batchInfo, d_scan_odata, d_ucntResult);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

#endif

}

void dtohTransfer(/*host resource*/unsigned int *h_ucntResult, unsigned int *h_result, batchInfo bi, /*device resource*/unsigned int *d_ucntResult, unsigned int *d_result)
{
	CUDA_SAFE_CALL(hipMemcpy(h_ucntResult, d_ucntResult, sizeof(unsigned int) * bi.queryNum, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, sizeof(unsigned int) * h_ucntResult[bi.queryNum - 1], hipMemcpyDeviceToHost));
}

void releaseResource(/*host resource*/unsigned int **h_constant, unsigned int **h_result, unsigned int **h_ucntResult, uint16_t **h_queryID_perBlock, unsigned char **h_bloom, regresstion_info_t **h_pSLinearInfoDetail, float **h_fLinearInfo, /*device resource*/unsigned int **d_lists, unsigned int **d_isCommon, unsigned int **d_scan_odata, unsigned int **d_result, unsigned int **d_ucntResult, uint16_t **d_queryID_perBlock, unsigned char **d_bloom, unsigned int **d_batchInfo, float **d_fLinearInfo)
{
	free(*h_result);
	*h_result = NULL;
	free(*h_ucntResult);
	*h_ucntResult = NULL;
	CUDA_SAFE_CALL(hipHostFree(*h_queryID_perBlock));
	*h_queryID_perBlock = NULL;
	CUDA_SAFE_CALL(hipHostFree(*h_constant));
	*h_constant = NULL;
	CUDA_SAFE_CALL(hipHostFree(*h_fLinearInfo));
	*h_fLinearInfo = NULL;
	free(*h_bloom);
	*h_bloom = NULL;
	/*
	free(*h_pSLinearInfoDetail);
	*h_pSLinearInfoDetail = NULL;
	*/


	CUDA_SAFE_CALL(hipFree(*d_lists));
	*d_lists = NULL;
	CUDA_SAFE_CALL(hipFree(*d_isCommon));
	*d_isCommon = NULL;
	CUDA_SAFE_CALL(hipFree(*d_scan_odata));
	*d_scan_odata = NULL;
	CUDA_SAFE_CALL(hipFree(*d_result));
	*d_result = NULL;
	CUDA_SAFE_CALL(hipFree(*d_ucntResult));
	*d_ucntResult = NULL;
	CUDA_SAFE_CALL(hipFree(*d_queryID_perBlock));
	*d_queryID_perBlock = NULL;
	CUDA_SAFE_CALL(hipFree(*d_bloom));
	*d_bloom = NULL;
	CUDA_SAFE_CALL(hipFree(*d_batchInfo));
	*d_batchInfo = NULL;
	/*
	CUDA_SAFE_CALL(hipFree(*d_fLinearInfo));
	*d_fLinearInfo = NULL;
	*/


	CUDPPResult res = cudppDestroyPlan(scanplan);
	if (CUDPP_SUCCESS != res)
	{
		printf("Error destroying CUDPPPlan\n");
		exit(1);
	}

	// by ysharp: release cudpplibrary	
	cudppDestroy(theCudpp);
}

unsigned int queryID = 0;
FILE *fpResNum = NULL;
void verify(batchInfo bi, unsigned int *h_ucntResult, unsigned int *h_result, checkSum *cs, vector<singleKeyword_t> *pvSSingleKey, unsigned int *h_lists)
{
	//local variables
	unsigned int i, j;	//loop controller
	unsigned int  nValidQueryIndex = 0;
	unsigned int resultNum;	//no more than at_trunc_count
	vector<singleKeyword_t>::iterator irSingleKey = pvSSingleKey->begin();
	irSingleKey = pvSSingleKey->begin();
	//local variables end

	char buf[256];

	for (i = 0; i < bi.nTotalQueryNum; ++i) {
		if (nValidQueryIndex) {
			if (irSingleKey != pvSSingleKey->end() && i == irSingleKey->nQueryIDInBatch) {
				resultNum = irSingleKey->nLen;
				cs->check_sum2 += resultNum;
				cs->check_sum3 ^= ((h_lists + irSingleKey->nOffset)[0]);
				cs->check_sum3 ^= ((h_lists + irSingleKey->nOffset + resultNum - 1)[0]);
				cs->check_sum3 ^= ((h_lists + irSingleKey->nOffset + resultNum / 2)[0]);

				irSingleKey++;
			}
			else if(h_ucntResult[nValidQueryIndex] - h_ucntResult[nValidQueryIndex - 1]) {
				resultNum = h_ucntResult[nValidQueryIndex] - h_ucntResult[nValidQueryIndex - 1];
				cs->check_sum2 += resultNum;
				cs->check_sum3 ^= (h_result[h_ucntResult[nValidQueryIndex - 1]]);
				cs->check_sum3 ^= (h_result[h_ucntResult[nValidQueryIndex - 1] + resultNum - 1]);
				cs->check_sum3 ^= (h_result[h_ucntResult[nValidQueryIndex - 1] + resultNum / 2]);
				nValidQueryIndex++;

				sprintf(buf, "%u,%u\n", queryID++, resultNum);
				fputs(buf, fpResNum);
			}
			else {
				cs->check_sum1++;
				nValidQueryIndex++;
				queryID++;
			}
		}
		else {
			if (irSingleKey != pvSSingleKey->end() && i == irSingleKey->nQueryIDInBatch) {
				resultNum = irSingleKey->nLen;
				cs->check_sum2 += resultNum;
				cs->check_sum3 ^= ((h_lists + irSingleKey->nOffset)[0]);
				cs->check_sum3 ^= ((h_lists + irSingleKey->nOffset + resultNum - 1)[0]);
				cs->check_sum3 ^= ((h_lists + irSingleKey->nOffset + resultNum / 2)[0]);

				irSingleKey++;
			}
			else if (h_ucntResult[nValidQueryIndex]) {
				resultNum = h_ucntResult[nValidQueryIndex];
				cs->check_sum2 += resultNum;
				cs->check_sum3 ^= (h_result[0]);
				cs->check_sum3 ^= (h_result[resultNum - 1]);
				cs->check_sum3 ^= (h_result[resultNum / 2]);
				nValidQueryIndex++;

				sprintf(buf, "%u,%u\n", queryID++, resultNum);
				fputs(buf, fpResNum);
			}
			else {
				cs->check_sum1++;
				nValidQueryIndex++;
				queryID++;
			}
		}
	}
}

//function
FILE *fpRes = NULL;
void Run(at_search_ind_t *patind, void *userdata, char *ptr_in, char *ptr_end)
{
	//local variables
	batchInfo bi;
	checkSum cs;
	vector<singleKeyword_t> vSSingleKey;
	char *ptr_in_old = ptr_in;
	char *ptr_end_old = ptr_end;
	//local ends


	hipEvent_t start, stop;
	float time_CPU = 0, time_htod = 0, time_kernel = 0, time_dtoh = 0;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	allocateResource(&h_constant, &h_queryID_perBlock, &h_result, &h_ucntResult, &(patind->m_pind2), &h_bloom, &h_pSLinearInfoDetail, &h_fLinearInfo, &d_queryID_perBlock, &d_lists, &d_isCommon, &d_scan_odata, &d_result, &d_ucntResult, patind->sz_ind2, &d_bloom, &d_batchInfo, &d_fLinearInfo, &config);

	//	DOCID_LIMIT = 30 * 0 * THREAD_NUM; 
	for (DOCID_LIMIT = 32 * 128 * THREAD_NUM; DOCID_LIMIT <= 32 * 128 * THREAD_NUM; DOCID_LIMIT *= 2)
	{
		for (int iteration = 0; iteration < ITERATION; ++iteration)
		{
			memset(&cs, 0, sizeof(struct checkSum));


			time_CPU = 0;
			time_htod = 0;
			time_dtoh = 0;
			time_kernel = 0;

			unsigned int nShortestSum = 0;


			printf("begin calculating...\n");
			unsigned int nThreshold = DOCID_LIMIT / 1024;
			printf("===========%uk   THREAD_NUM %u==========\n", nThreshold, THREAD_NUM);

			unsigned int batchID = 0;
			unsigned int nValidQueryNum = 0;
			unsigned int nTotalQueryNum = 0;
			ptr_in = ptr_in_old;
			ptr_end = ptr_end_old;
			while (ptr_in < ptr_end /*&& batchID < 1024*/)
			{
				hipEventRecord(start, 0);

				bi = CPUPreprocess(h_constant, h_queries, h_startBlockId, h_queriesOffset, h_queryID_perBlock, h_pSLinearInfoDetail, h_fLinearInfo, patind, &ptr_in, &ptr_end, &vSSingleKey);
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_CPU += elapsedTime;


				hipEventRecord(start, 0);

				htodTransfer(h_constant, h_queryID_perBlock, h_fLinearInfo, d_batchInfo, d_queryID_perBlock, bi, d_batchInfo, d_fLinearInfo);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_htod += elapsedTime;


				hipEventRecord(start, 0);

				kernelInvoke(bi, d_lists, d_isCommon, d_scan_odata, d_result, d_ucntResult, d_queryID_perBlock, d_batchInfo, d_fLinearInfo);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_kernel += elapsedTime;

#ifdef fullProcess
				hipEventRecord(start, 0);

				dtohTransfer(h_ucntResult, h_result, bi, d_ucntResult, d_result);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				time_dtoh += elapsedTime; 

				verify(bi, h_ucntResult, h_result, &cs, &vSSingleKey, (unsigned int *)(patind->m_pind2));
#endif


#ifdef debug
				//debug; print batch info
				printf("---------batch Info ----------\n"); 
				printf("batchID: %u\n", batchID);
				printf("queryNum:%u\tblockNum:%u\tconstantUsed:%u\tshortestSum:%u\n", bi.queryNum, bi.blockNum, bi.constantUsedInByte, bi.ucntShortest_sum);
				printf("first resultNum: ======%u======\n", h_ucntResult[0]);
				printf("---------batch Info ends-----\n");
				//debug ends
#endif

				batchID++;
				nValidQueryNum += bi.queryNum;
				nTotalQueryNum += bi.nTotalQueryNum;
				nShortestSum += bi.ucntShortest_sum;

				/*
				//response and throughput
				char buf[256];
				double dResponse = (cur_time_CPU + cur_time_htod + cur_time_kernel + cur_time_dtoh);
				sprintf(buf, "%lf\n", dResponse);
				fputs(buf, fpRes);
				fflush(fpRes);
				//ends
				*/

			}

			printf("total batch Num:%u\tshortestSum: %u\tvalidQueryNum:%u\tBFQueryNum:%u\n", batchID, nShortestSum, nValidQueryNum, nBFQueryNum);
			//print time
			printf("CPU time:%lf ms\nhtod transfer time:%lf ms\n===kernel time===:%lf ms\ndtoh transfer time:%lf ms\ntotal time: %lf\n", time_CPU, time_htod, time_kernel, time_dtoh, time_CPU + time_htod + time_kernel + time_dtoh);


			//print check_sum; for baidu:16392f 22203,2743515,4174671
			printf("************************\nchecksum:%x\ncheck_sum1:%u\tcheck_sum2: %u\tcheck_sum3:%u\n************************\n", cs.check_sum1 ^ cs.check_sum2 ^ cs.check_sum3, cs.check_sum1, cs.check_sum2, cs.check_sum3);

			//response and throughput
			char buf[256];
			printf("totalQueryNum: %u\n", nTotalQueryNum);
			double dResponse = (time_CPU + time_htod + time_kernel + time_dtoh) / ((double)batchID);
			double dThroughput = ((double)nTotalQueryNum) / (time_CPU + time_htod + time_kernel + time_dtoh) * 1000;
			//sprintf(buf, "%lf\t%lf\n", dResponse, dThroughput);
			/*sprintf(buf, "%lf\n", dResponse);*/
			sprintf(buf, "%uk\t%lf\t%lf\t%lf\t%lf\t%lf\n", nThreshold, dResponse, dThroughput, time_CPU, time_kernel, time_htod + time_dtoh);
			fputs(buf, fpRes);
			fflush(fpRes);
			//ends

			printf("res: %lf\tthroughput: %lf\n", dResponse, dThroughput);
			printf("########### resNum: %u\tredundancy rate: %lf\tactualFalsePositive: %lf #############\n", cs.check_sum2, (cs.check_sum2 - GOV_RESNUM) / ((double)GOV_RESNUM), (cs.check_sum2 - GOV_RESNUM) / ((double)nShortestSum));
			//delete timers
			//delete timers end
		}

		printf("\n");
	}

	releaseResource(&h_constant, &h_result, &h_ucntResult, &h_queryID_perBlock, &h_bloom, &h_pSLinearInfoDetail, &h_fLinearInfo, &d_lists, &d_isCommon, &d_scan_odata, &d_result, &d_ucntResult, &d_queryID_perBlock, &d_bloom, &d_batchInfo, &d_fLinearInfo);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

// Do some free operations
void terminator(){
	if(ptr!=NULL) free(ptr);
	if(patind->m_pind1!=NULL) free(patind->m_pind1);
	if(patind->m_pind2!=NULL) free(patind->m_pind2);
	if(patind!=NULL) free(patind);
}


void runTest (int argc, char **argv) {
	printf("program start\n");
	if (argc != 2) {
		std::cout << "wrong number of arguments" << std::endl;
		exit(1);
	}

	string dataset = argv[1];
	string index_dir = "/media/gov2disk/naiyong/dataset/" + dataset + "/";

	string input_file = index_dir + dataset + ".query";
	at_search_ind_t *at_ind = NULL;

	FILE* fp = fopen(input_file.c_str(),"rb");
	if (!fp)
	{
		printf("at_test_data open failed\terr code:%u\n", errno);
		exit(1);
	}

	cout << "reading " << input_file << endl;
	struct stat buf;
	stat(input_file.c_str(), &buf);  
	int size = buf.st_size;  //????test?Ĵ?С
	ptr = (unsigned char*) malloc(size);;  //????test??ָ??
	readFile(ptr,fp);
	at_ind = as_load_atind(index_dir.c_str(), dataset.c_str());
	fclose(fp);


	//CUT_DEVICE_INIT(argc, argv);
	if (argc > 2)
	{
		printf("%s\n", argv[2]);
		CUDA_SAFE_CALL(hipSetDevice(strtol(argv[2], NULL, 10)));
	}
	else //default set to Tesla
	{
		CUDA_SAFE_CALL(hipSetDevice(0));
	}



	fpRes = fopen("PPOPP_RES_BINARY.txt", "w+");
	fpResNum = fopen("resNumFromVerify.txt", "w+");

	//run
	Run(at_ind, 0, (char *)ptr, (char *)ptr + buf.st_size);


	fclose(fpRes);
	fclose(fpResNum);

	terminator();
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
    runTest( argc, argv);
	hipDeviceReset();	
}
