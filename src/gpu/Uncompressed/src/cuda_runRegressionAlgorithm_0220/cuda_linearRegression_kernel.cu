#include "hip/hip_runtime.h"
#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdint.h>
#include <stdio.h>
#include <cudpp.h>

#define SDATA( index)      cutilBankChecker(sdata, index)

// ind1
typedef struct _at_term_ind1 {
	uint32_t m_urlcount;  // length 
	uint64_t m_off;       // offset (in bytes)
} at_term_ind1_t;


typedef struct _at_search_ind {
	FILE* fd_ind1;				//һ�������ļ����
	FILE* fd_ind2;				//���������ļ����
	uint64_t sz_ind1;			//һ�������ļ���С
	uint64_t sz_ind2;			//���������ļ���С
	at_term_ind1_t *m_pind1;	//һ������ָ��
	unsigned char *m_pind2;		//��������ָ��
	int m_tcount;				//AT�����е�term����
} at_search_ind_t;


#define	MAX_QUERY_LEN	80							//public

// query
typedef struct _query_input {
	uint32_t tnum;
	uint32_t tno[MAX_QUERY_LEN];
} query_input_t;


// note: we assume that ind2 is less than 16GB
//       so that offset can fit in 32-bit
struct testCell {
	uint32_t tcount;                // number of terms
	uint32_t ucnt[MAX_QUERY_LEN];   // lengths
	uint32_t uwlist[MAX_QUERY_LEN]; // offsets (in 4bytes) 
	uint32_t tno[MAX_QUERY_LEN];    // term ids
}; 


// cudpp scan
CUDPPConfiguration config;
CUDPPHandle scanplan = 0;;
CUDPPHandle theCudpp = 0;




//filled by CPUPreprocess
struct batchInfo
{
	//index info in constant memory
	//unsigned int startOffset_queriesOffset;
	//unsigned int startOffset_queries;
	//index info in constant memory ends

	unsigned int blockNum;	//block number needed by kernel
	unsigned int queryNum;	//query number contained by the batch
	unsigned int constantUsedInByte;	//capacity used in constant memory
	unsigned int ucntShortest_sum;	//length sum of shortest lists, which is the actual docID number in d_isCommon and d_scan_odata
	unsigned int nTotalQueryNum;  //including those queries have only one keyword
	unsigned int nLinearInfoInByte;  //bytes d_fLinearInfo needs to transfer
};

struct checkSum
{
	unsigned int check_sum1;	//number of queries with empty result
	unsigned int check_sum2;	//number of docIDs in valid result
	unsigned int check_sum3;
};

//global constants
#define QUERY_NUM 16184  //һ�����ٸ���ѯ
#define	TOTAL_QUERY 33337  //��33337��һЩ
#define THREAD_NUM 256 
#define ROUTE_BINARY 0
#define ROUTE_BLOOM 1
#define GOV_RESNUM 183430377 
#define BAIDU_RESNUM 2743515
unsigned int DOCID_LIMIT = 30 * 2 * THREAD_NUM;
const unsigned int at_trunc_count = 300000;  //�ٶ�Լ���Ľ�����޸���
const unsigned int baseSize = sizeof(unsigned int) * at_trunc_count * 16;  //���Ͱ���͵����ޣ���Ϊ�����Ǿ�̬������Դ
const unsigned int batchInfoElementNum = sizeof(struct batchInfo) / sizeof(unsigned int);	//element number in batchInfo structure
//global constants end


// Definintion

#define	MAX_PATH_LEN	1024
#define MAX_SUBQUERY_TERM 80
#define MAX_QUERY_NUM 512

typedef struct _singleKeyword
{
	unsigned int nLen, nOffset, nQueryIDInBatch;
} singleKeyword_t;

typedef struct _regression_info
{
	//regression formula
	float fSlope;
	float fIntercept;
	float fRSquare;
	float fMultiple;  //the multiple of distance and VDistance; equals to sqrt(1 + fSlope * fSlope)
	//two parallel lines
	float fInterceptHigh;
	float fInterceptLow;
	//horizontal range
	unsigned int nRangeLeft;
	unsigned int nRangeRight;
	//x-axis multiple
	unsigned int nMagnification;
} regresstion_info_t;

//global device variables
__constant__ unsigned int d_constantOld[4];  //constant space
unsigned int *d_lists;  //equivalent to d_ind2
unsigned int *d_isCommon;  //0-1 array
unsigned int *d_scan_odata;  //for scan operation
unsigned int *d_result;  //storage results, used in compact operation
unsigned int *d_ucntResult;	//number of results per query
uint16_t *d_queryID_perBlock;	//corresponding to queryID for each block 
unsigned char *d_bloom;
unsigned int *d_batchInfo;
float *d_fLinearInfo;  //slope..., intercept..., lRange..., rRange... 
//global device variables end

//global host variables
//unsigned int h_constant[163800];  //corresponding to d_constant
unsigned int *h_constant;  //corresponding to d_constant
unsigned int *h_result;  //corresponding to d_result
unsigned int *h_lists;	//corresponding to d_lists
unsigned int *h_ucntResult;	//corresponding to d_ucntResult
uint16_t *h_queryID_perBlock;	//corresponding to queryID for each block
unsigned char *h_bloom;
float *h_fLinearInfo;  //slope..., intercept..., lRange..., rRange... 
regresstion_info_t *h_pSLinearInfoDetail;  //stores at_regression_info file

unsigned int h_queries[16380];	//these three arrays compose the entire content of constant memory
unsigned int h_startBlockId[16380];
unsigned int h_queriesOffset[16380];
unsigned int h_baseOffset[16380];

//global host variables end


__global__ void zero(int* isCommon,int size)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int subscript = blockDim.x * blockIdx.x + tid;
	if(subscript >= size) 	return ;

	isCommon[subscript] = 0;
}


__global__ void mqSearch(unsigned int *d_constant, unsigned int *d_lists, unsigned int *d_isCommon, uint16_t *d_queryID_perBlock, float *d_fLinearInfo)
{
	//shared
	__shared__ unsigned int s_array[128];
	__shared__ float s_linearInfo[160];
	__shared__ unsigned int activeThreadNum;	//thread number needed in the block
	__shared__ unsigned int startIdx;	//start index of the shortest list in the query for this block
	__shared__ unsigned int s_tcount;	//tcount in shared
	__shared__ unsigned int constantOffset;	//segment offset from the head of constant memory
	//shared ends

	//registers
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int i, j;	//loop control variables
	__shared__ unsigned int queryNum;
	unsigned int startBlockId = 0;	//the first block ID who deals with the query
	unsigned int queryID = 0;	//my queryID
	unsigned int queriesOffset = 0;
	unsigned int tcount;	//number of terms
	unsigned int *isCommon;
	unsigned int *ucnts, *uwlists;
	//registers end


	if (tid == 0)
	{
		//thread0, fetch queryNum from the first segment of constant memory
		queryNum = ((batchInfo*)d_constant)->queryNum;

		//search in startBlockId segment, confirm which query I will deal with
		constantOffset = batchInfoElementNum;

		//fetch the result of reduction; and set the queryID and startBlockId
		queryID = d_queryID_perBlock[bid]; //queryNum - s_array[0] - 1;
		startBlockId = (d_constant + constantOffset)[queryID];

		//calculate startIdx
		startIdx = (blockIdx.x - startBlockId) * THREAD_NUM;

		//get the query offset in queries segment
		constantOffset += queryNum;
		queriesOffset = (d_constant + constantOffset)[queryID];
		constantOffset += queryNum + queriesOffset;

		//get tcount
		s_tcount = (d_constant + constantOffset)[0];
		constantOffset += 1;

	}
	__syncthreads();


	tcount = s_tcount;

	//retrieve baseOffset, ucnts and uwlists
	//s_array[0] is baseOffset
	if (tid < (tcount << 1) + 3)
	{
		s_array[tid] = (d_constant + constantOffset)[tid];
	}
	__syncthreads();
	
	//set my own ucnts and wulists
	ucnts = s_array + 3;
	uwlists = ucnts + tcount;
	isCommon = d_isCommon + s_array[0];

	//calculate activeThreadNum
	//set shared baseOffset
	if (threadIdx.x == 0)
	{
		activeThreadNum = ucnts[0] - startIdx >= blockDim.x ? blockDim.x : ucnts[0] - startIdx;
	}
	__syncthreads();

	//retrieve linearInfo
	if (tid < tcount * 4)
	{
		s_linearInfo[tid] = d_fLinearInfo[s_array[1] + tid];
	}
	__syncthreads();


	//binary search
	int eleIndex = startIdx + threadIdx.x;
	int shortestStart = uwlists[0];
	int found = 0;	//identify whether has been found
	int listIdx = 0;
	int leftNew = 0, rightNew = 0;
	int leftNewTmp = 0, rightNewTmp = 0;
	int iPivotTmp = 0, nListIdxTmp = 0;

	
	
	//get left and right borders
	__shared__ unsigned int s_nPHead , s_nPTail ;
	__shared__ unsigned int s_Border[80];  //as the max number of lists per query is 36
	if (0 == tid)
	{
		s_nPHead = (d_lists[shortestStart + eleIndex]);
		s_nPTail = (d_lists[shortestStart + eleIndex + activeThreadNum - 1]);
	}
	__syncthreads();

	if (tid < (tcount << 1))
	{
		nListIdxTmp = tid >> 1;
//		if (ucnts[nListIdxTmp] > 1)
		{
			/*if (tid % 2)  //right border*/
			if (tid & 1)  //right border
			{
				/*iPivotTmp = ((float)s_nPTail - s_linearInfo[tcount + nListIdxTmp]) / s_linearInfo[nListIdxTmp];*/
				iPivotTmp = __fdividef(((float)s_nPTail - s_linearInfo[tcount + nListIdxTmp]), s_linearInfo[nListIdxTmp]);
				/*iPivotTmp = __fdiv_rn(((float)s_nPTail - s_linearInfo[tcount + nListIdxTmp]), s_linearInfo[nListIdxTmp]);*/
				/*if (iPivotTmp < 0) iPivotTmp = 0;*/
				/*rightNewTmp = iPivotTmp + (int)s_linearInfo[tcount * 3 + nListIdxTmp];*/
				rightNewTmp = iPivotTmp + (int)s_linearInfo[__umul24(tcount, 3) + nListIdxTmp];
				if (rightNewTmp < ucnts[nListIdxTmp])
				{
					s_Border[tid] = rightNewTmp;
				}
				else
				{
					s_Border[tid] = ucnts[nListIdxTmp] - 1;
				}
			}
			else  //left border
			{
				/*iPivotTmp = ((float)s_nPHead- s_linearInfo[tcount + nListIdxTmp]) / s_linearInfo[nListIdxTmp];*/
				iPivotTmp = __fdividef(((float)s_nPHead- s_linearInfo[tcount + nListIdxTmp]), s_linearInfo[nListIdxTmp]);
				/*iPivotTmp = __fdiv_rn(((float)s_nPHead- s_linearInfo[tcount + nListIdxTmp]), s_linearInfo[nListIdxTmp]);*/
				/*if (iPivotTmp < 0) iPivotTmp = 0;*/
				/*leftNewTmp = iPivotTmp - (int)s_linearInfo[tcount * 2 + nListIdxTmp];*/
				leftNewTmp = iPivotTmp - (int)s_linearInfo[__umul24(tcount , 2) + nListIdxTmp];
				if (leftNewTmp > 0)
				{
					s_Border[tid] = leftNewTmp;
				}
				else
				{
					s_Border[tid] = 0;
				}
			}
		}
	}
	__syncthreads();
	//get ends



	if (threadIdx.x < activeThreadNum)
	{
		int middle, left, right;
		unsigned int p, q;
		unsigned int listLen = 0;
		int nPivot = 0;	//the index calculated from regression formula
		p = (d_lists[shortestStart + eleIndex]);
		isCommon[eleIndex] = 1;	//set to 1 first

		for (listIdx = 1; listIdx < tcount; ++listIdx)
		{
			listLen = ucnts[listIdx];
			left = 0; right = listLen - 1;
			found = 0;

			//for less formula calculation
			if (ucnts[listIdx] > 1)
			{
				left = s_Border[listIdx << 1];
				right = s_Border[(listIdx <<  1) + 1];
			}



			/*
			if (ucnts[listIdx] > 1)
			{
				nPivot = __fdividef((((float)p) - s_linearInfo[tcount + listIdx]) , s_linearInfo[listIdx]);
				leftNew = nPivot - (int)(s_linearInfo[__umul24(tcount , 2) + listIdx]); 
				rightNew = nPivot + s_linearInfo[__umul24(tcount , 3) + listIdx]; 
				if (leftNew > left)
				{
					left = leftNew;
				}
				if (rightNew < right)
				{
					right = rightNew;
				}
			}
			*/
			
			/*
			if (ucnts[listIdx] > 1)
			{
				nPivot = (((float)p) - s_linearInfo[tcount + listIdx]) / s_linearInfo[listIdx];

				if (nPivot <= 0 || nPivot >= listLen)
				{
					leftNew = nPivot - (int)(s_linearInfo[tcount * 2 + listIdx]); 
					rightNew = nPivot + s_linearInfo[tcount * 3 + listIdx]; 
					if (leftNew > left)
					{
						left = leftNew;
					}
					if (rightNew < right)
					{
						right = rightNew;
					}
				}
				else
				{
					if (p > (d_lists[uwlists[listIdx] + nPivot]))
					{
						rightNew = nPivot + s_linearInfo[tcount * 3 + listIdx]; 
						if (rightNew < right)
						{
							right = rightNew;
						}
						leftNew = nPivot  > 0 ? nPivot  : 0;
						left = leftNew;
					}
					else
					{
						leftNew = nPivot - (int)(s_linearInfo[tcount * 2 + listIdx]); 
						if (leftNew > left)
						{
							left = leftNew;
						}
						//rightNew = nPivot < right ? nPivot : right;
						//right = rightNew;
						rightNew = nPivot > 0 ? nPivot : right;
						right = rightNew;
					}
				}
			}
		*/
//			isCommon[eleIndex] = right;//s_linearInfo[listIdx + tcount];//(d_lists[uwlists[listIdx] + 7995]);


			while (left <= right)
			{
				middle = (left + right) >> 1;
				q = (d_lists[uwlists[listIdx] + middle]);

				if (p == q)
				{
					found = 1;
					break;
				}
				if (p > q)
				{
					left = middle + 1;
				}
				else
				{
					right = middle - 1;
				}
			};

			if (!found)
			{
				isCommon[eleIndex] = 0;
				break;
			}
		}
	}



	__syncthreads();
}


__global__ void saCompact(unsigned int *d_constant, unsigned int *d_lists, unsigned int* d_isCommon, unsigned int *d_scan_odata, unsigned int *d_result, uint16_t *d_queryID_perBlock)
{
	//shared
	__shared__ unsigned int uwlist; 
	__shared__ unsigned int ucnt;
	__shared__ unsigned int activeThreadNum;	//thread number needed in the block
	__shared__ unsigned int startIdx;	//start index of the shortest list in the query for this block
	__shared__ unsigned int constantOffset;	//offset to the head of constant memory, as pointer to constant memory is not allowed
	__shared__ unsigned int baseOffset;
	__shared__ unsigned int s_tcount;
	//shared ends

	//registers
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int i, j;	//loop control variables
	__shared__ unsigned int queryNum;
	unsigned int startBlockId = 0;	//the first block ID who deals with the query
	unsigned int queryID = 0;	//my queryID
	unsigned int queriesOffset = 0;
	unsigned int *isCommon, *scan_odata;
	//registers end


	if (threadIdx.x == 0)
	{
		//thread0, fetch queryNum from the first segment of constant memory
		queryNum = ((batchInfo*)d_constant)->queryNum;

		//search in startBlockId segment, confirm which query I will deal with
		constantOffset = batchInfoElementNum;

		//fetch the queryID from d_queryID_per_block, and set startBlockId
		queryID = d_queryID_perBlock[bid];
		startBlockId = (d_constant + constantOffset)[queryID];

		//calculate startIdx
		startIdx = (blockIdx.x - startBlockId) * THREAD_NUM;

		//get the query offset in queries segment
		constantOffset += queryNum;
		queriesOffset = (d_constant + constantOffset)[queryID];
		constantOffset += queryNum + queriesOffset;

		s_tcount = *(d_constant + constantOffset);
		baseOffset = *(d_constant + constantOffset + 1);
		constantOffset += 4;
		
		//retieve detailed lists info
		ucnt = ((d_constant + constantOffset))[0];
		uwlist = ((d_constant + constantOffset + s_tcount))[0];
	}
	__syncthreads();


	//calculate activeThreadNum
	if (threadIdx.x == 0)
	{
		activeThreadNum = ucnt - startIdx >= blockDim.x ? blockDim.x : ucnt - startIdx;
	}
	__syncthreads();


	//compact
	unsigned int eleIndex = startIdx + threadIdx.x;
	unsigned int shortestStart = uwlist;
	isCommon = d_isCommon + baseOffset;
	scan_odata = d_scan_odata + baseOffset;



	if (threadIdx.x < activeThreadNum)
	{
		if (isCommon[eleIndex])
		{
			d_result[scan_odata[eleIndex] - 1] = d_lists[shortestStart + eleIndex];
		}
	}
	__syncthreads();
}

__global__ void ucntResult(unsigned int *d_constant, unsigned int *d_scan_odata, unsigned int *d_ucntResult)
{
	//shared
	__shared__ unsigned int queryNum;
	//shared ends

	//registers
	unsigned int queryID = blockDim.x * blockIdx.x + threadIdx.x;	//each thread deals with one query
	unsigned int baseOffset;
	unsigned int *segPointer = d_constant;	//for baseOffset segment in constant memory
	//registers end

	if (threadIdx.x == 0)
	{
		queryNum = ((batchInfo*)d_constant)->queryNum;
	}
	__syncthreads();

	if (queryID < queryNum - 1)
	{
		//segPointer += batchInfoElementNum + queryNum;
		//baseOffset = segPointer[queryID + 1];
		segPointer += batchInfoElementNum + queryNum;
		segPointer += queryNum + segPointer[queryID + 1];
		baseOffset = segPointer[1];
		d_ucntResult[queryID] = d_scan_odata[baseOffset - 1];
	}
	else if (queryNum - 1 == queryID)
	{
		baseOffset = ((batchInfo*)d_constant)->ucntShortest_sum;
		d_ucntResult[queryID] = d_scan_odata[baseOffset - 1];
	}
	__syncthreads();
}


#endif // #ifndef _TEMPLATE_KERNEL_H_


